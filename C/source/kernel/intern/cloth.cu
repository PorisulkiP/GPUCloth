#include "hip/hip_runtime.h"
#include "MEM_guardedalloc.cuh"

#include "cloth_types.cuh"
#include "mesh_types.h"
#include "meshdata_types.cuh"
#include "object_types.cuh"

#include "edgehash.h"
#include "linklist.cuh"
#include "B_math.h"
#include "rand.h"
#include "utildefines.h"
#include "mesh_runtime.h"

#include "DEG_depsgraph.h"
#include "DEG_depsgraph_query.h"

#include "bvhutils.h"
#include "cloth.h"
#include "effect.h"
#include "pointcache.h"
#include "task.hh"

#include "SIM_mass_spring.cuh"

typedef struct BendSpringRef {
  int index;
  int polys;
  ClothSpring *spring;
} BendSpringRef;

/******************************************************************************
 *
 * External interface called by modifier.c clothModifier functions.
 *
 ******************************************************************************/

static BVHTree *bvhtree_build_from_cloth(ClothModifierData *clmd, float epsilon)
{
  if (!clmd) {
	return NULL;
  }

  Cloth *cloth = clmd->clothObject;

  if (!cloth) {
	return NULL;
  }

  ClothVertex *verts = cloth->verts;
  const MVertTri *vt = cloth->tri;

  /* in the moment, return zero if no faces there */
  if (!cloth->primitive_num) {
	return NULL;
  }

  /* create quadtree with k=26 */
  BVHTree *bvhtree = BLI_bvhtree_new(cloth->primitive_num, epsilon, 4, 26);

  /* fill tree */
  if (clmd->hairdata == NULL) {
	for (int i = 0; i < cloth->primitive_num; i++, vt++) {
	  float co[3][3];

	  copy_v3_v3(co[0], verts[vt->tri[0]].xold);
	  copy_v3_v3(co[1], verts[vt->tri[1]].xold);
	  copy_v3_v3(co[2], verts[vt->tri[2]].xold);

	  BLI_bvhtree_insert(bvhtree, i, co[0], 3);
	}
  }
  else {
	MEdge *edges = cloth->edges;

	for (int i = 0; i < cloth->primitive_num; i++) {
	  float co[2][3];

	  copy_v3_v3(co[0], verts[edges[i].v1].xold);
	  copy_v3_v3(co[1], verts[edges[i].v2].xold);

	  BLI_bvhtree_insert(bvhtree, i, co[0], 2);
	}
  }

  /* balance tree */
  BLI_bvhtree_balance(bvhtree);

  return bvhtree;
}

void bvhtree_update_from_cloth(ClothModifierData *clmd, bool moving, bool self)
{
  uint i = 0;
  Cloth *cloth = clmd->clothObject;
  BVHTree *bvhtree;
  ClothVertex *verts = cloth->verts;
  const MVertTri *vt;

  BLI_assert(!(clmd->hairdata != NULL && self));

  if (self) {
	bvhtree = cloth->bvhselftree;
  }
  else {
	bvhtree = cloth->bvhtree;
  }

  if (!bvhtree) {
	return;
  }

  vt = cloth->tri;

  /* update vertex position in bvh tree */
  if (clmd->hairdata == NULL) {
	if (verts && vt) {
	  for (i = 0; i < cloth->primitive_num; i++, vt++) {
		float co[3][3], co_moving[3][3];
		bool ret;

		/* copy new locations into array */
		if (moving) {
		  copy_v3_v3(co[0], verts[vt->tri[0]].txold);
		  copy_v3_v3(co[1], verts[vt->tri[1]].txold);
		  copy_v3_v3(co[2], verts[vt->tri[2]].txold);

		  /* update moving positions */
		  copy_v3_v3(co_moving[0], verts[vt->tri[0]].tx);
		  copy_v3_v3(co_moving[1], verts[vt->tri[1]].tx);
		  copy_v3_v3(co_moving[2], verts[vt->tri[2]].tx);

		  ret = BLI_bvhtree_update_node(bvhtree, i, co[0], co_moving[0], 3);
		}
		else {
		  copy_v3_v3(co[0], verts[vt->tri[0]].tx);
		  copy_v3_v3(co[1], verts[vt->tri[1]].tx);
		  copy_v3_v3(co[2], verts[vt->tri[2]].tx);

		  ret = BLI_bvhtree_update_node(bvhtree, i, co[0], NULL, 3);
		}

		/* check if tree is already full */
		if (ret == false) {
		  break;
		}
	  }

	  BLI_bvhtree_update_tree(bvhtree);
	}
  }
  else {
	if (verts) {
	  MEdge *edges = cloth->edges;

	  for (i = 0; i < cloth->primitive_num; i++) {
		float co[2][3];

		copy_v3_v3(co[0], verts[edges[i].v1].tx);
		copy_v3_v3(co[1], verts[edges[i].v2].tx);

		if (!BLI_bvhtree_update_node(bvhtree, i, co[0], NULL, 2)) {
		  break;
		}
	  }

	  BLI_bvhtree_update_tree(bvhtree);
	}
  }
}

// ��������� ������ �����
int do_step_cloth(Depsgraph *depsgraph, Object *ob, ClothModifierData *clmd, Mesh *result, int framenr)
{
  Cloth *cloth = clmd->clothObject;
  ClothVertex* verts = cloth->verts;
  MVert *mvert = result->mvert;

  /* ������������� ������� ��� ������������ ������� � �� ������������ ��������������. */
  for (uint i = 0; i < clmd->clothObject->mvert_num; ++i, ++verts)
  {
	// save the previous position.
	// ���������� ���������� �������.
	copy_v3_v3(verts->xold, verts->xconst);
	copy_v3_v3(verts->txold, verts->x);

	// Get the current position.
	// ��������� ������� �������.
	copy_v3_v3(verts->xconst, mvert[i].co);
	mul_m4_v3(ob->obmat, verts->xconst);
  }
  // ��������� ��������, ������� ����� ������ �� �����
  // ���� nullptr
  ListBase* effectors = effectors_create(depsgraph, ob, NULL, clmd->sim_parms->effector_weights, false);

  if (clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_DYNAMIC_BASEMESH) 
  {
	// ����������� ������ ������ �� ClothModifierData* clmd � Mesh *result
	cloth_update_verts(ob, clmd, result);
  }

  if ((clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_DYNAMIC_BASEMESH) ||
	  (clmd->sim_parms->vgroup_shrink > 0) || (clmd->sim_parms->shrink_min != 0.0f)) 
  {
	cloth_update_spring_lengths(clmd, result);
  }

  cloth_update_springs(clmd);

  // ��������� ������� ���������
  // TIMEIT_START(cloth_step)

  /* call the solver. */
  if (SIM_cloth_solve(depsgraph, ob, framenr, clmd, effectors))
  {
	  return 1;
  }
  else
  {
	  return 0;
  }

  // TIMEIT_END(cloth_step)
  // printf ( "%f\n", ( float ) tval() );
}

/************************************************
 * clothModifier_do - main simulation function
 ************************************************/
Cloth* clothModifier_do(ClothModifierData *clmd, Depsgraph *depsgraph, Object *ob, Mesh *mesh)
{
	PointCache *cache;
	PTCacheID pid;
	float timescale = 1;
	int framenr = 0, startframe = 0, endframe = 0, cache_result = 0;

	framenr = depsgraph->ctime;
	cache = clmd->point_cache;
	// ���� ��������� ���� ������, ��� �������� � �����, �� ��������� ��� �� ����� �����
	if (framenr > endframe)  framenr = endframe;

	BKE_ptcache_id_from_cloth(&pid, ob, clmd);
	BKE_ptcache_id_time(&pid, depsgraph->scene, framenr, &startframe, &endframe, &timescale);
	clmd->sim_parms->timescale = timescale * clmd->sim_parms->time_scale;

	// ���� � ���������� ������ "��������" ��� ���-�� ����� � ������� ��������� � ����� �� �����, �� �� ���������
	if (clmd->sim_parms->reset || (clmd->clothObject && mesh->totvert != clmd->clothObject->mvert_num)) 
	{
		clmd->sim_parms->reset = 0;
		cache->flag |= PTCACHE_OUTDATED;
		BKE_ptcache_id_reset(depsgraph->scene, &pid, PTCACHE_RESET_OUTDATED);
		BKE_ptcache_validate(cache, 0);
		cache->last_exact = 0;
		cache->flag &= ~PTCACHE_REDO_NEEDED;
	}

	// ��������� ���������� ������ � ������� ������������� �������
	if (framenr < startframe) 
	{
		BKE_ptcache_invalidate(cache);
		return nullptr;
	}

	// ���� ���� ������ � ������ �����
	if (framenr == startframe) 
	{
		BKE_ptcache_id_reset(depsgraph->scene, &pid, PTCACHE_RESET_OUTDATED);
		BKE_ptcache_validate(cache, framenr);
		cache->flag &= ~PTCACHE_REDO_NEEDED;
		clmd->clothObject->last_frame = framenr;
		return nullptr;
	}

  // try to read from cache
  // ������� ��������� �� ����, ���� �� ����� ���� �������
	bool can_simulate = false;// (framenr == clmd->clothObject->last_frame + 1) && !(cache->flag & PTCACHE_BAKED);

  //cache_result = BKE_ptcache_read(&pid, (float)framenr + depsgraph->scene->r.subframe, can_simulate);

  //if (cache_result == PTCACHE_READ_EXACT || cache_result == PTCACHE_READ_INTERPOLATED ||
  //    (!can_simulate && cache_result == PTCACHE_READ_OLD)) {
  //  SIM_cloth_solver_set_positions(clmd);
  //  BKE_ptcache_validate(cache, framenr);

  //  if (cache_result == PTCACHE_READ_INTERPOLATED && cache->flag & PTCACHE_REDO_NEEDED) 
  //  {
  //    BKE_ptcache_write(&pid, framenr);
  //  }

  //  clmd->clothObject->last_frame = framenr;

  //  return;
  //}
  //if (cache_result == PTCACHE_READ_OLD) {
  //  SIM_cloth_solver_set_positions(clmd);
  //}
  //else if (
  //    /* 2.4x disabled lib, but this can be used in some cases, testing further - campbell */
  //    /*ob->id.lib ||*/ (cache->flag & PTCACHE_BAKED)) {
  //  /* if baked and nothing in cache, do nothing */
  //  BKE_ptcache_invalidate(cache);
  //  return;
  //}

  /* if on second frame, write cache for first frame */
  //if (cache->simframe == startframe && (cache->flag & PTCACHE_OUTDATED || cache->last_exact == 0)) 
  //{
  //  BKE_ptcache_write(&pid, startframe);
  //}

	clmd->sim_parms->timescale = 1;//*= framenr - cache->simframe;

	//BKE_ptcache_validate(cache, framenr);

	/* do simulation */
	if (do_step_cloth(depsgraph, ob, clmd, mesh, framenr)) 
	{
		// ���� ��������� ������ �������, �� ��������� ���
		//BKE_ptcache_write(&pid, framenr);
	}
	else {
		// � ��������� ������ �� ��������
		BKE_ptcache_invalidate(cache);
	}
	clmd->clothObject->last_frame = framenr;
	return clmd->clothObject;
}

/* frees all */
void cloth_free_modifier(ClothModifierData *clmd)
{
  Cloth *cloth = NULL;

  if (!clmd) { return; }

  cloth = clmd->clothObject;

  if (cloth) {
	SIM_cloth_solver_free(clmd);

	/* Free the verts. */
	if (cloth->verts != NULL) {
	  MEM_freeN(cloth->verts);
	}

	cloth->verts = NULL;
	cloth->mvert_num = 0;

	/* Free the springs. */
	if (cloth->springs != NULL) {
	  LinkNode *search = cloth->springs;
	  while (search) {
		ClothSpring *spring = (ClothSpring*)search->link;

		MEM_SAFE_FREE(spring->pa);
		MEM_SAFE_FREE(spring->pb);

		MEM_freeN(spring);
		search = search->next;
	  }
	  BLI_linklist_free(cloth->springs, NULL);

	  cloth->springs = NULL;
	}

	cloth->springs = NULL;
	cloth->numsprings = 0;

	/* free BVH collision tree */
	if (cloth->bvhtree) {
	  BLI_bvhtree_free(cloth->bvhtree);
	}

	if (cloth->bvhselftree) {
	  BLI_bvhtree_free(cloth->bvhselftree);
	}

	/* we save our faces for collision objects */
	if (cloth->tri) {
	  MEM_freeN(cloth->tri);
	}

	if (cloth->edgeset) {
	  BLI_edgeset_free(cloth->edgeset);
	}

	if (cloth->sew_edge_graph) {
	  BLI_edgeset_free(cloth->sew_edge_graph);
	  cloth->sew_edge_graph = NULL;
	}
	MEM_freeN(cloth);
	clmd->clothObject = NULL;
  }
}

void cloth_free_modifier_extern(ClothModifierData* clmd)
{
	Cloth* cloth = NULL;
	if (!clmd) { return; }

	cloth = clmd->clothObject;

	if (cloth) {
		SIM_cloth_solver_free(clmd);

		/* Free the verts. */
		MEM_SAFE_FREE(cloth->verts);
		cloth->mvert_num = 0;

		/* Free the springs. */
		if (cloth->springs != NULL) {
			LinkNode* search = cloth->springs;
			while (search) {
				ClothSpring* spring = (ClothSpring*)search->link;

				MEM_SAFE_FREE(spring->pa);
				MEM_SAFE_FREE(spring->pb);

				MEM_freeN(spring);
				search = search->next;
			}
			BLI_linklist_free(cloth->springs, NULL);

			cloth->springs = NULL;
		}

		cloth->springs = NULL;
		cloth->numsprings = 0;

		/* free BVH collision tree */
		if (cloth->bvhtree) {
			BLI_bvhtree_free(cloth->bvhtree);
		}

		if (cloth->bvhselftree) {
			BLI_bvhtree_free(cloth->bvhselftree);
		}

		/* we save our faces for collision objects */
		if (cloth->tri) {
			MEM_freeN(cloth->tri);
		}

		if (cloth->edgeset) {
			BLI_edgeset_free(cloth->edgeset);
		}

		if (cloth->sew_edge_graph) {
			BLI_edgeset_free(cloth->sew_edge_graph);
			cloth->sew_edge_graph = NULL;
		}
		MEM_freeN(cloth);
		clmd->clothObject = NULL;
	}
}

int cloth_uses_vgroup(ClothModifierData *clmd)
{
  return (((clmd->coll_parms->flags & CLOTH_COLLSETTINGS_FLAG_SELF) &&
		   (clmd->coll_parms->vgroup_selfcol > 0)) ||
		  ((clmd->coll_parms->flags & CLOTH_COLLSETTINGS_FLAG_ENABLED) &&
		   (clmd->coll_parms->vgroup_objcol > 0)) ||
		  (clmd->sim_parms->vgroup_pressure > 0) || (clmd->sim_parms->vgroup_struct > 0) ||
		  (clmd->sim_parms->vgroup_bend > 0) || (clmd->sim_parms->vgroup_shrink > 0) ||
		  (clmd->sim_parms->vgroup_intern > 0) || (clmd->sim_parms->vgroup_mass > 0));
}

static float cloth_shrink_factor(ClothModifierData *clmd, ClothVertex *verts, int i1, int i2)
{
  /* Linear interpolation between min and max shrink factor based on weight. */
  float base = 1.0f - clmd->sim_parms->shrink_min;
  float shrink_factor_delta = clmd->sim_parms->shrink_min - clmd->sim_parms->shrink_max;

  float k1 = base + shrink_factor_delta * verts[i1].shrink_factor;
  float k2 = base + shrink_factor_delta * verts[i2].shrink_factor;

  /* Use geometrical mean to average two factors since it behaves better
   * for diagonals when a rectangle transforms into a trapezoid. */
  return sqrtf(k1 * k2);
}
/* -------------------------------------------------------------------- */
/** \name Spring Network Building Implementation
 * \{ */

 void spring_verts_ordered_set(ClothSpring *spring, int v0, int v1)
{
  if (v0 < v1) {
	spring->ij = v0;
	spring->kl = v1;
  }
  else {
	spring->ij = v1;
	spring->kl = v0;
  }
}

static void cloth_free_edgelist(LinkNodePair *edgelist, uint mvert_num)
{
  if (edgelist) {
	for (uint i = 0; i < mvert_num; i++) {
	  BLI_linklist_free(edgelist[i].list, NULL);
	}

	MEM_freeN(edgelist);
  }
}

static void cloth_free_errorsprings(Cloth *cloth,
									LinkNodePair *edgelist,
									BendSpringRef *spring_ref)
{
  if (cloth->springs != NULL) {
	LinkNode *search = cloth->springs;
	while (search) {
	  ClothSpring *spring = (ClothSpring*)search->link;

	  MEM_SAFE_FREE(spring->pa);
	  MEM_SAFE_FREE(spring->pb);

	  MEM_freeN(spring);
	  search = search->next;
	}
	BLI_linklist_free(cloth->springs, NULL);

	cloth->springs = NULL;
  }

  cloth_free_edgelist(edgelist, cloth->mvert_num);

  MEM_SAFE_FREE(spring_ref);

  if (cloth->edgeset) {
	BLI_edgeset_free(cloth->edgeset);
	cloth->edgeset = NULL;
  }
}

 void cloth_bend_poly_dir(
	ClothVertex *verts, int i, int j, const int *inds, int len, float r_dir[3])
{
  float cent[3] = {0};
  float fact = 1.0f / len;

  for (int x = 0; x < len; x++) {
	madd_v3_v3fl(cent, verts[inds[x]].xrest, fact);
  }

  normal_tri_v3(r_dir, verts[i].xrest, verts[j].xrest, cent);
}

static float cloth_spring_angle(
	ClothVertex *verts, int i, int j, int *i_a, int *i_b, int len_a, int len_b)
{
  float dir_a[3], dir_b[3];
  float tmp[3], vec_e[3];
  float sin, cos;

  /* Poly vectors. */
  cloth_bend_poly_dir(verts, j, i, i_a, len_a, dir_a);
  cloth_bend_poly_dir(verts, i, j, i_b, len_b, dir_b);

  /* Edge vector. */
  sub_v3_v3v3(vec_e, verts[i].xrest, verts[j].xrest);
  normalize_v3(vec_e);

  /* Compute angle. */
  cos = dot_v3v3(dir_a, dir_b);

  cross_v3_v3v3(tmp, dir_a, dir_b);
  sin = dot_v3v3(tmp, vec_e);

  return atan2f(sin, cos);
}

static void cloth_hair_update_bending_targets(ClothModifierData *clmd)
{
  Cloth *cloth = clmd->clothObject;
  LinkNode *search = NULL;
  float hair_frame[3][3], dir_old[3], dir_new[3];
  int prev_mn; /* to find hair chains */

  if (!clmd->hairdata) {
	return;
  }

  /* XXX Note: we need to propagate frames from the root up,
   * but structural hair springs are stored in reverse order.
   * The bending springs however are then inserted in the same
   * order as vertices again ...
   * This messy situation can be resolved when solver data is
   * generated directly from a dedicated hair system.
   */

  prev_mn = -1;
  for (search = cloth->springs; search; search = search->next) {
	ClothSpring *spring = (ClothSpring*)search->link;
	ClothHairData *hair_ij, *hair_kl;
	bool is_root = spring->kl != prev_mn;

	if (spring->type != CLOTH_SPRING_TYPE_BENDING_HAIR) {
	  continue;
	}

	hair_ij = &clmd->hairdata[spring->ij];
	hair_kl = &clmd->hairdata[spring->kl];
	if (is_root) {
	  /* initial hair frame from root orientation */
	  copy_m3_m3(hair_frame, hair_ij->rot);
	  /* surface normal is the initial direction,
	   * parallel transport then keeps it aligned to the hair direction
	   */
	  copy_v3_v3(dir_new, hair_frame[2]);
	}

	copy_v3_v3(dir_old, dir_new);
	sub_v3_v3v3(dir_new, cloth->verts[spring->mn].x, cloth->verts[spring->kl].x);
	normalize_v3(dir_new);

	/* get local targets for kl/mn vertices by putting rest targets into the current frame,
	 * then multiply with the rest length to get the actual goals
	 */

	mul_v3_m3v3(spring->target, hair_frame, hair_kl->rest_target);
	mul_v3_fl(spring->target, spring->restlen);

	/* move frame to next hair segment */
	cloth_parallel_transport_hair_frame(hair_frame, dir_old, dir_new);

	prev_mn = spring->mn;
  }
}

static void cloth_hair_update_bending_rest_targets(ClothModifierData *clmd)
{
  Cloth *cloth = clmd->clothObject;
  LinkNode *search = NULL;
  float hair_frame[3][3], dir_old[3], dir_new[3];
  int prev_mn; /* to find hair roots */

  if (!clmd->hairdata) {
	return;
  }

  /* XXX Note: we need to propagate frames from the root up,
   * but structural hair springs are stored in reverse order.
   * The bending springs however are then inserted in the same
   * order as vertices again ...
   * This messy situation can be resolved when solver data is
   * generated directly from a dedicated hair system.
   */

  prev_mn = -1;
  for (search = cloth->springs; search; search = search->next) {
	ClothSpring *spring = (ClothSpring*)search->link;
	ClothHairData *hair_ij, *hair_kl;
	bool is_root = spring->kl != prev_mn;

	if (spring->type != CLOTH_SPRING_TYPE_BENDING_HAIR) {
	  continue;
	}

	hair_ij = &clmd->hairdata[spring->ij];
	hair_kl = &clmd->hairdata[spring->kl];
	if (is_root) {
	  /* initial hair frame from root orientation */
	  copy_m3_m3(hair_frame, hair_ij->rot);
	  /* surface normal is the initial direction,
	   * parallel transport then keeps it aligned to the hair direction
	   */
	  copy_v3_v3(dir_new, hair_frame[2]);
	}

	copy_v3_v3(dir_old, dir_new);
	sub_v3_v3v3(dir_new, cloth->verts[spring->mn].xrest, cloth->verts[spring->kl].xrest);
	normalize_v3(dir_new);

	/* dir expressed in the hair frame defines the rest target direction */
	copy_v3_v3(hair_kl->rest_target, dir_new);
	mul_transposed_m3_v3(hair_frame, hair_kl->rest_target);

	/* move frame to next hair segment */
	cloth_parallel_transport_hair_frame(hair_frame, dir_old, dir_new);

	prev_mn = spring->mn;
  }
}

/* update stiffness if vertex group values are changing from frame to frame */
static void cloth_update_springs(ClothModifierData *clmd)
{
  Cloth *cloth = clmd->clothObject;
  LinkNode *search = NULL;

  search = cloth->springs;
  while (search) {
	ClothSpring *spring = (ClothSpring*)search->link;

	spring->lin_stiffness = 0.0f;

	if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) {
	  if (spring->type & CLOTH_SPRING_TYPE_BENDING) {
		spring->ang_stiffness = (cloth->verts[spring->kl].bend_stiff +
								 cloth->verts[spring->ij].bend_stiff) /
								2.0f;
	  }
	}

	if (spring->type & CLOTH_SPRING_TYPE_STRUCTURAL) {
	  spring->lin_stiffness = (cloth->verts[spring->kl].struct_stiff +
							   cloth->verts[spring->ij].struct_stiff) /
							  2.0f;
	}
	else if (spring->type & CLOTH_SPRING_TYPE_SHEAR) {
	  spring->lin_stiffness = (cloth->verts[spring->kl].shear_stiff +
							   cloth->verts[spring->ij].shear_stiff) /
							  2.0f;
	}
	else if (spring->type == CLOTH_SPRING_TYPE_BENDING) {
	  spring->lin_stiffness = (cloth->verts[spring->kl].bend_stiff +
							   cloth->verts[spring->ij].bend_stiff) /
							  2.0f;
	}
	else if (spring->type & CLOTH_SPRING_TYPE_INTERNAL) {
	  spring->lin_stiffness = (cloth->verts[spring->kl].internal_stiff +
							   cloth->verts[spring->ij].internal_stiff) /
							  2.0f;
	}
	else if (spring->type == CLOTH_SPRING_TYPE_BENDING_HAIR) {
	  ClothVertex *v1 = &cloth->verts[spring->ij];
	  ClothVertex *v2 = &cloth->verts[spring->kl];
	  if (clmd->hairdata) {
		/* copy extra hair data to generic cloth vertices */
		v1->bend_stiff = clmd->hairdata[spring->ij].bending_stiffness;
		v2->bend_stiff = clmd->hairdata[spring->kl].bending_stiffness;
	  }
	  spring->lin_stiffness = (v1->bend_stiff + v2->bend_stiff) / 2.0f;
	}
	else if (spring->type == CLOTH_SPRING_TYPE_GOAL) {
	  /* Warning: Appending NEW goal springs does not work
	   * because implicit solver would need reset! */

	  /* Activate / Deactivate existing springs */
	  if ((!(cloth->verts[spring->ij].flags & CLOTH_VERT_FLAG_PINNED)) &&
		  (cloth->verts[spring->ij].goal > ALMOST_ZERO)) {
		spring->flags &= ~CLOTH_SPRING_FLAG_DEACTIVATE;
	  }
	  else {
		spring->flags |= CLOTH_SPRING_FLAG_DEACTIVATE;
	  }
	}

	search = search->next;
  }

  cloth_hair_update_bending_targets(clmd);
}

// Update rest verts, for dynamically deformable cloth
// �������� ��������� ������� ��� ����������� ������������� �����
static void cloth_update_verts(Object *ob, ClothModifierData *clmd, Mesh *mesh)
{
  MVert *mvert = mesh->mvert;
  ClothVertex *verts = clmd->clothObject->verts;

  // vertex count is already ensured to match
  // ���������� ������ ��� ���������� ��� ������������
  for (uint i = 0; i < mesh->totvert; ++i, ++verts)
  {
	copy_v3_v3(verts->xrest, mvert[i].co);
	mul_m4_v3(ob->obmat, verts->xrest);
  }
}

/* Write rest vert locations to a copy of the mesh. */
static Mesh *cloth_make_rest_mesh(ClothModifierData *clmd, Mesh *mesh)
{
  Mesh* new_mesh = NULL;// BKE_mesh_copy_for_eval(mesh, false);
  ClothVertex *verts = clmd->clothObject->verts;
  MVert *mvert = new_mesh->mvert;

  /* vertex count is already ensured to match */
  for (unsigned i = 0; i < mesh->totvert; i++, verts++) {
	copy_v3_v3(mvert[i].co, verts->xrest);
  }

  return new_mesh;
}

/* Update spring rest length, for dynamically deformable cloth */
static void cloth_update_spring_lengths(ClothModifierData *clmd, Mesh *mesh)
{
  Cloth *cloth = clmd->clothObject;
  LinkNode *search = cloth->springs;
  uint struct_springs = 0;
  uint i = 0;
  uint mvert_num = (uint)mesh->totvert;
  float shrink_factor;

  clmd->sim_parms->avg_spring_len = 0.0f;

  for (i = 0; i < mvert_num; i++) {
	cloth->verts[i].avg_spring_len = 0.0f;
  }

  while (search) {
	ClothSpring *spring = (ClothSpring*)search->link;

	if (spring->type != CLOTH_SPRING_TYPE_SEWING) {
	  if (spring->type & (CLOTH_SPRING_TYPE_STRUCTURAL | CLOTH_SPRING_TYPE_SHEAR |
						  CLOTH_SPRING_TYPE_BENDING | CLOTH_SPRING_TYPE_INTERNAL)) {
		shrink_factor = cloth_shrink_factor(clmd, cloth->verts, spring->ij, spring->kl);
	  }
	  else {
		shrink_factor = 1.0f;
	  }

	  spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest, cloth->verts[spring->ij].xrest) *
						shrink_factor;

	  if (spring->type & CLOTH_SPRING_TYPE_BENDING) {
		spring->restang = cloth_spring_angle(
			cloth->verts, spring->ij, spring->kl, spring->pa, spring->pb, spring->la, spring->lb);
	  }
	}

	if (spring->type & CLOTH_SPRING_TYPE_STRUCTURAL) {
	  clmd->sim_parms->avg_spring_len += spring->restlen;
	  cloth->verts[spring->ij].avg_spring_len += spring->restlen;
	  cloth->verts[spring->kl].avg_spring_len += spring->restlen;
	  struct_springs++;
	}

	search = search->next;
  }

  if (struct_springs > 0) {
	clmd->sim_parms->avg_spring_len /= struct_springs;
  }

  for (i = 0; i < mvert_num; i++) {
	if (cloth->verts[i].spring_count > 0) {
	  cloth->verts[i].avg_spring_len = cloth->verts[i].avg_spring_len * 0.49f /
									   ((float)cloth->verts[i].spring_count);
	}
  }
}

 void cross_identity_v3(float r[3][3], const float v[3])
{
  zero_m3(r);
  r[0][1] = v[2];
  r[0][2] = -v[1];
  r[1][0] = -v[2];
  r[1][2] = v[0];
  r[2][0] = v[1];
  r[2][1] = -v[0];
}

 void madd_m3_m3fl(float r[3][3], const float m[3][3], float f)
{
  r[0][0] += m[0][0] * f;
  r[0][1] += m[0][1] * f;
  r[0][2] += m[0][2] * f;
  r[1][0] += m[1][0] * f;
  r[1][1] += m[1][1] * f;
  r[1][2] += m[1][2] * f;
  r[2][0] += m[2][0] * f;
  r[2][1] += m[2][1] * f;
  r[2][2] += m[2][2] * f;
}

void cloth_parallel_transport_hair_frame(float mat[3][3],
										 const float dir_old[3],
										 const float dir_new[3])
{
  float rot[3][3];

  /* rotation between segments */
  rotation_between_vecs_to_mat3(rot, dir_old, dir_new);

  /* rotate the frame */
  mul_m3_m3m3(mat, rot, mat);
}

/* Add a shear and a bend spring between two verts within a poly. */
static bool cloth_add_shear_bend_spring(ClothModifierData *clmd,
										LinkNodePair *edgelist,
										const MLoop *mloop,
										const MPoly *mpoly,
										int i,
										int j,
										int k)
{
  Cloth *cloth = clmd->clothObject;
  ClothSpring *spring;
  const MLoop *tmp_loop;
  float shrink_factor;
  int x, y;

  /* Combined shear/bend properties. */
  spring = (ClothSpring *)MEM_callocN(sizeof(ClothSpring), "cloth spring");

  if (!spring) {
	return false;
  }

  spring_verts_ordered_set(
	  spring, mloop[mpoly[i].loopstart + j].v, mloop[mpoly[i].loopstart + k].v);

  shrink_factor = cloth_shrink_factor(clmd, cloth->verts, spring->ij, spring->kl);
  spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest, cloth->verts[spring->ij].xrest) *
					shrink_factor;
  spring->type |= CLOTH_SPRING_TYPE_SHEAR;
  spring->lin_stiffness = (cloth->verts[spring->kl].shear_stiff +
						   cloth->verts[spring->ij].shear_stiff) /
						  2.0f;

  if (edgelist) {
	BLI_linklist_append(&edgelist[spring->ij], spring);
	BLI_linklist_append(&edgelist[spring->kl], spring);
  }

  /* Bending specific properties. */
  if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) {
	spring->type |= CLOTH_SPRING_TYPE_BENDING;

	spring->la = k - j + 1;
	spring->lb = mpoly[i].totloop - k + j + 1;

	spring->pa = (int*)MEM_mallocN(sizeof(*spring->pa) * spring->la, "spring poly");
	if (!spring->pa) {
	  return false;
	}

	spring->pb = (int*)MEM_mallocN(sizeof(*spring->pb) * spring->lb, "spring poly");
	if (!spring->pb) {
	  return false;
	}

	tmp_loop = mloop + mpoly[i].loopstart;

	for (x = 0; x < spring->la; x++) {
	  spring->pa[x] = tmp_loop[j + x].v;
	}

	for (x = 0; x <= j; x++) {
	  spring->pb[x] = tmp_loop[x].v;
	}

	for (y = k; y < mpoly[i].totloop; x++, y++) {
	  spring->pb[x] = tmp_loop[y].v;
	}

	spring->mn = -1;

	spring->restang = cloth_spring_angle(
		cloth->verts, spring->ij, spring->kl, spring->pa, spring->pb, spring->la, spring->lb);

	spring->ang_stiffness = (cloth->verts[spring->ij].bend_stiff +
							 cloth->verts[spring->kl].bend_stiff) /
							2.0f;
  }

  BLI_linklist_prepend(&cloth->springs, spring);

  return true;
}

 bool cloth_bend_set_poly_vert_array(int **poly, int len, const MLoop *mloop)
{
  int *p = (int*)MEM_mallocN(sizeof(int) * len, "spring poly");

  if (!p) {
	return false;
  }

  for (int i = 0; i < len; i++, mloop++) {
	p[i] = mloop->v;
  }

  *poly = p;

  return true;
}

static bool find_internal_spring_target_vertex(BVHTreeFromMesh *treedata,
											   uint v_idx,
											   RNG *rng,
											   float max_length,
											   float max_diversion,
											   bool check_normal,
											   uint *r_tar_v_idx)
{
  float co[3], no[3], new_co[3];
  float radius;

  copy_v3_v3(co, treedata->vert[v_idx].co);
  normal_short_to_float_v3(no, treedata->vert[v_idx].no);
  negate_v3(no);

  float vec_len = sin(max_diversion);
  float offset[3];

  offset[0] = 0.5f - BLI_rng_get_float(rng);
  offset[1] = 0.5f - BLI_rng_get_float(rng);
  offset[2] = 0.5f - BLI_rng_get_float(rng);

  normalize_v3(offset);
  mul_v3_fl(offset, vec_len);
  add_v3_v3(no, offset);
  normalize_v3(no);

  /* Nudge the start point so we do not hit it with the ray. */
  copy_v3_v3(new_co, no);
  mul_v3_fl(new_co, FLT_EPSILON);
  add_v3_v3(new_co, co);

  radius = 0.0f;
  if (max_length == 0.0f) {
	max_length = FLT_MAX;
  }

  BVHTreeRayHit rayhit = {0};
  rayhit.index = -1;
  rayhit.dist = max_length;

  BLI_bvhtree_ray_cast(
	  treedata->tree, new_co, no, radius, &rayhit, treedata->raycast_callback, treedata);

  uint vert_idx = -1;
  const MLoop *mloop = treedata->loop;
  const MLoopTri *lt = NULL;

  if (rayhit.index != -1 && rayhit.dist <= max_length) {
	if (check_normal && dot_v3v3(rayhit.no, no) < 0.0f) {
	  /* We hit a point that points in the same direction as our starting point. */
	  return false;
	}

	float min_len = FLT_MAX;
	lt = &treedata->looptri[rayhit.index];

	for (int i = 0; i < 3; i++) {
	  uint tmp_vert_idx = mloop[lt->tri[i]].v;
	  if (tmp_vert_idx == v_idx) {
		/* We managed to hit ourselves. */
		return false;
	  }

	  float len = len_v3v3(co, rayhit.co);
	  if (len < min_len) {
		min_len = len;
		vert_idx = tmp_vert_idx;
	  }
	}

	*r_tar_v_idx = vert_idx;
	return true;
  }

  return false;
}

bool cloth_build_springs(ClothModifierData *clmd, Mesh *mesh)
{
	Cloth *cloth = clmd->clothObject;
	ClothSpring *spring = NULL, *tspring = NULL, *tspring2 = NULL;
	uint struct_springs = 0, shear_springs = 0, bend_springs = 0, struct_springs_real = 0;
	uint mvert_num = (uint)mesh->totvert;
	uint numedges = (uint)mesh->totedge;
	uint numpolys = (uint)mesh->totpoly;
	float shrink_factor;
	const MEdge *medge = mesh->medge;
	const MPoly *mpoly = mesh->mpoly;
	const MLoop *mloop = mesh->mloop;
	int index2 = 0; /* our second vertex index */
	LinkNodePair *edgelist = nullptr;
	EdgeSet *edgeset = nullptr;
	LinkNode *search = nullptr, *search2 = nullptr;
	BendSpringRef *spring_ref = nullptr;

	/* error handling */
	if (numedges == 0) 
	{
		printf("Error handling\n");
		return false;
	}

  /* NOTE: handling ownership of springs and edgeset is quite sloppy
   * currently they are never initialized but assert just to be sure */
  BLI_assert(cloth->springs == NULL);
  BLI_assert(cloth->edgeset == NULL);

  cloth->springs = NULL;
  cloth->edgeset = NULL;

  if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) 
  {
	spring_ref = (BendSpringRef*)MEM_callocN(sizeof(*spring_ref) * numedges, "temp bend spring reference");

	if (!spring_ref) 
	{
	  return false;
	}
  }
  else 
  {
	edgelist = (LinkNodePair*)MEM_callocN(sizeof(*edgelist) * mvert_num, "cloth_edgelist_alloc");

	if (!edgelist) 
	{
	  return false;
	}
  }

  bool use_internal_springs = (clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_INTERNAL_SPRINGS);

  if (use_internal_springs && numpolys > 0) 
  {
	BVHTreeFromMesh treedata = {NULL};
	uint tar_v_idx;
	Mesh *tmp_mesh = nullptr;
	RNG *rng;

	/* If using the rest shape key, it's necessary to make a copy of the mesh. */
	if (clmd->sim_parms->shapekey_rest &&
		!(clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_DYNAMIC_BASEMESH)) {
	  tmp_mesh = cloth_make_rest_mesh(clmd, mesh);
	  //BKE_mesh_calc_normals(tmp_mesh);
	}

	EdgeSet *existing_vert_pairs = BLI_edgeset_new("cloth_sewing_edges_graph");
	BKE_bvhtree_from_mesh_get(&treedata, tmp_mesh ? tmp_mesh : mesh, BVHTREE_FROM_LOOPTRI, 2);
	rng = BLI_rng_new_srandom(0);

	for (int i = 0; i < mvert_num; i++) {
	  if (find_internal_spring_target_vertex(
			  &treedata,
			  i,
			  rng,
			  clmd->sim_parms->internal_spring_max_length,
			  clmd->sim_parms->internal_spring_max_diversion,
			  (clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_INTERNAL_SPRINGS_NORMAL),
			  &tar_v_idx)) {
		if (BLI_edgeset_haskey(existing_vert_pairs, i, tar_v_idx)) {
		  /* We have already created a spring between these verts! */
		  continue;
		}

		BLI_edgeset_insert(existing_vert_pairs, i, tar_v_idx);

		spring = (ClothSpring *)MEM_callocN(sizeof(ClothSpring), "cloth spring");

		if (spring) {
		  spring_verts_ordered_set(spring, i, tar_v_idx);

		  shrink_factor = cloth_shrink_factor(clmd, cloth->verts, spring->ij, spring->kl);
		  spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest,
									 cloth->verts[spring->ij].xrest) *
							shrink_factor;
		  spring->lin_stiffness = (cloth->verts[spring->kl].internal_stiff +
								   cloth->verts[spring->ij].internal_stiff) /
								  2.0f;
		  spring->type = CLOTH_SPRING_TYPE_INTERNAL;

		  spring->flags = 0;

		  BLI_linklist_prepend(&cloth->springs, spring);

		  if (spring_ref) {
			spring_ref[i].spring = spring;
		  }
		}
		else {
		  cloth_free_errorsprings(cloth, edgelist, spring_ref);
		  BLI_edgeset_free(existing_vert_pairs);
		  free_bvhtree_from_mesh(&treedata);
		  //if (tmp_mesh) { BKE_mesh_free(tmp_mesh); }
		  return false;
		}
	  }
	}
	BLI_edgeset_free(existing_vert_pairs);
	free_bvhtree_from_mesh(&treedata);
	//if (tmp_mesh) { BKE_mesh_free(tmp_mesh); }
	BLI_rng_free(rng);
  }

  clmd->sim_parms->avg_spring_len = 0.0f;
  for (int i = 0; i < mvert_num; i++) {
	cloth->verts[i].avg_spring_len = 0.0f;
  }

  if (clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_SEW) {
	/* cloth->sew_edge_graph should not exist before this */
	BLI_assert(cloth->sew_edge_graph == NULL);
	cloth->sew_edge_graph = BLI_edgeset_new("cloth_sewing_edges_graph");
  }

  /* Structural springs. */
  for (int i = 0; i < numedges; i++) {
	spring = (ClothSpring *)MEM_callocN(sizeof(ClothSpring), "cloth spring");

	if (spring) {
	  spring_verts_ordered_set(spring, medge[i].v1, medge[i].v2);
	  if (clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_SEW && medge[i].flag & ME_LOOSEEDGE) {
		/* handle sewing (loose edges will be pulled together) */
		spring->restlen = 0.0f;
		spring->lin_stiffness = 1.0f;
		spring->type = CLOTH_SPRING_TYPE_SEWING;

		BLI_edgeset_insert(cloth->sew_edge_graph, medge[i].v1, medge[i].v2);
	  }
	  else {
		shrink_factor = cloth_shrink_factor(clmd, cloth->verts, spring->ij, spring->kl);
		spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest,
								   cloth->verts[spring->ij].xrest) *
						  shrink_factor;
		spring->lin_stiffness = (cloth->verts[spring->kl].struct_stiff +
								 cloth->verts[spring->ij].struct_stiff) /
								2.0f;
		spring->type = CLOTH_SPRING_TYPE_STRUCTURAL;

		clmd->sim_parms->avg_spring_len += spring->restlen;
		cloth->verts[spring->ij].avg_spring_len += spring->restlen;
		cloth->verts[spring->kl].avg_spring_len += spring->restlen;
		cloth->verts[spring->ij].spring_count++;
		cloth->verts[spring->kl].spring_count++;
		struct_springs_real++;
	  }

	  spring->flags = 0;
	  struct_springs++;

	  BLI_linklist_prepend(&cloth->springs, spring);

	  if (spring_ref) {
		spring_ref[i].spring = spring;
	  }
	}
	else {
	  cloth_free_errorsprings(cloth, edgelist, spring_ref);
	  return false;
	}
  }

  if (struct_springs_real > 0) {
	clmd->sim_parms->avg_spring_len /= struct_springs_real;
  }

  for (int i = 0; i < mvert_num; i++) {
	if (cloth->verts[i].spring_count > 0) {
	  cloth->verts[i].avg_spring_len = cloth->verts[i].avg_spring_len * 0.49f /
									   ((float)cloth->verts[i].spring_count);
	}
  }

  edgeset = BLI_edgeset_new_ex(__func__, numedges);
  cloth->edgeset = edgeset;

  if (numpolys) {
	for (int i = 0; i < numpolys; i++) {
	  /* Shear springs. */
	  /* Triangle faces already have shear springs due to structural geometry. */
	  if (mpoly[i].totloop > 3) {
		for (int j = 1; j < mpoly[i].totloop - 1; j++) {
		  if (j > 1) {
			if (cloth_add_shear_bend_spring(clmd, edgelist, mloop, mpoly, i, 0, j)) {
			  shear_springs++;

			  if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) {
				bend_springs++;
			  }
			}
			else {
			  cloth_free_errorsprings(cloth, edgelist, spring_ref);
			  return false;
			}
		  }

		  for (int k = j + 2; k < mpoly[i].totloop; k++) {
			if (cloth_add_shear_bend_spring(clmd, edgelist, mloop, mpoly, i, j, k)) {
			  shear_springs++;

			  if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) {
				bend_springs++;
			  }
			}
			else {
			  cloth_free_errorsprings(cloth, edgelist, spring_ref);
			  return false;
			}
		  }
		}
	  }

	  /* Angular bending springs along struct springs. */
	  if (clmd->sim_parms->bending_model == CLOTH_BENDING_ANGULAR) {
		const MLoop *ml = mloop + mpoly[i].loopstart;

		for (int j = 0; j < mpoly[i].totloop; j++, ml++) {
		  BendSpringRef *curr_ref = &spring_ref[ml->e];
		  curr_ref->polys++;

		  /* First poly found for this edge, store poly index. */
		  if (curr_ref->polys == 1) {
			curr_ref->index = i;
		  }
		  /* Second poly found for this edge, add bending data. */
		  else if (curr_ref->polys == 2) {
			spring = curr_ref->spring;

			spring->type |= CLOTH_SPRING_TYPE_BENDING;

			spring->la = mpoly[curr_ref->index].totloop;
			spring->lb = mpoly[i].totloop;

			if (!cloth_bend_set_poly_vert_array(
					&spring->pa, spring->la, &mloop[mpoly[curr_ref->index].loopstart]) ||
				!cloth_bend_set_poly_vert_array(
					&spring->pb, spring->lb, &mloop[mpoly[i].loopstart])) {
			  cloth_free_errorsprings(cloth, edgelist, spring_ref);
			  return false;
			}

			spring->mn = ml->e;

			spring->restang = cloth_spring_angle(cloth->verts,
												 spring->ij,
												 spring->kl,
												 spring->pa,
												 spring->pb,
												 spring->la,
												 spring->lb);

			spring->ang_stiffness = (cloth->verts[spring->ij].bend_stiff +
									 cloth->verts[spring->kl].bend_stiff) /
									2.0f;

			bend_springs++;
		  }
		  /* Third poly found for this edge, remove bending data. */
		  else if (curr_ref->polys == 3) {
			spring = curr_ref->spring;

			spring->type &= ~CLOTH_SPRING_TYPE_BENDING;
			MEM_freeN(spring->pa);
			MEM_freeN(spring->pb);
			spring->pa = NULL;
			spring->pb = NULL;

			bend_springs--;
		  }
		}
	  }
	}

	/* Linear bending springs. */
	if (clmd->sim_parms->bending_model == CLOTH_BENDING_LINEAR) {
	  search2 = cloth->springs;

	  for (int i = struct_springs; i < struct_springs + shear_springs; i++) {
		if (!search2) {
		  break;
		}

		tspring2 = (ClothSpring*)search2->link;
		search = edgelist[tspring2->kl].list;

		while (search) {
		  tspring = (ClothSpring*)search->link;
		  index2 = ((tspring->ij == tspring2->kl) ? (tspring->kl) : (tspring->ij));

		  /* Check for existing spring. */
		  /* Check also if startpoint is equal to endpoint. */
		  if ((index2 != tspring2->ij) && !BLI_edgeset_haskey(edgeset, tspring2->ij, index2)) {
			spring = (ClothSpring *)MEM_callocN(sizeof(ClothSpring), "cloth spring");

			if (!spring) {
			  cloth_free_errorsprings(cloth, edgelist, spring_ref);
			  return false;
			}

			spring_verts_ordered_set(spring, tspring2->ij, index2);
			shrink_factor = cloth_shrink_factor(clmd, cloth->verts, spring->ij, spring->kl);
			spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest,
									   cloth->verts[spring->ij].xrest) *
							  shrink_factor;
			spring->type = CLOTH_SPRING_TYPE_BENDING;
			spring->lin_stiffness = (cloth->verts[spring->kl].bend_stiff +
									 cloth->verts[spring->ij].bend_stiff) /
									2.0f;
			BLI_edgeset_insert(edgeset, spring->ij, spring->kl);
			bend_springs++;

			BLI_linklist_prepend(&cloth->springs, spring);
		  }

		  search = search->next;
		}

		search2 = search2->next;
	  }
	}
  }
  else if (struct_springs > 2) {
	/* bending springs for hair strands
	 * The current algorithm only goes through the edges in order of the mesh edges list
	 * and makes springs between the outer vert of edges sharing a vertice. This works just
	 * fine for hair, but not for user generated string meshes. This could/should be later
	 * extended to work with non-ordered edges so that it can be used for general "rope
	 * dynamics" without the need for the vertices or edges to be ordered through the length
	 * of the strands. -jahka */
	 search = cloth->springs;
	 search2 = search->next;
	 while (search && search2) {
		 tspring = (ClothSpring*)search->link;
		 tspring2 = (ClothSpring*)search2->link;

		 if (tspring->ij == tspring2->kl) {
			 spring = (ClothSpring*)MEM_callocN(sizeof(ClothSpring), "cloth spring");

			 if (!spring) {
				 cloth_free_errorsprings(cloth, edgelist, spring_ref);
				 return false;
			 }

			 spring->ij = tspring2->ij;
			 spring->kl = tspring->kl;
			 spring->restlen = len_v3v3(cloth->verts[spring->kl].xrest,
				 cloth->verts[spring->ij].xrest);
			 spring->type = CLOTH_SPRING_TYPE_BENDING;
			 spring->lin_stiffness = (cloth->verts[spring->kl].bend_stiff +
				 cloth->verts[spring->ij].bend_stiff) /
				 2.0f;
			 bend_springs++;

			 BLI_linklist_prepend(&cloth->springs, spring);
		 }

		 search = search->next;
		 search2 = search2->next;
	}

	cloth_hair_update_bending_rest_targets(clmd);
  }

  /* note: the edges may already exist so run reinsert */

  /* insert other near springs in edgeset AFTER bending springs are calculated (for selfcolls) */
  for (int i = 0; i < numedges; i++) { /* struct springs */
	BLI_edgeset_add(edgeset, medge[i].v1, medge[i].v2);
  }

  for (int i = 0; i < numpolys; i++) { /* edge springs */
	if (mpoly[i].totloop == 4) {
	  BLI_edgeset_add(edgeset, mloop[mpoly[i].loopstart + 0].v, mloop[mpoly[i].loopstart + 2].v);
	  BLI_edgeset_add(edgeset, mloop[mpoly[i].loopstart + 1].v, mloop[mpoly[i].loopstart + 3].v);
	}
  }

  MEM_SAFE_FREE(spring_ref);

  cloth->numsprings = struct_springs + shear_springs + bend_springs;

  cloth_free_edgelist(edgelist, mvert_num);
  return true;
}


static bool cloth_from_object(Object* ob, ClothModifierData* clmd, Mesh* mesh, float UNUSED(framenr), int first)
{
	int i = 0;
	MVert* mvert = NULL;
	ClothVertex* verts = NULL;
	const float(*shapekey_rest)[3] = NULL;
	const float tnull[3] = { 0, 0, 0 };

	/* If we have a clothObject, free it. */
	if (clmd->clothObject != NULL) 
	{
		cloth_free_modifier(clmd);
	}

	/* Allocate a new cloth object. */
	clmd->clothObject = (Cloth*)MEM_callocN(sizeof(Cloth), "cloth");
	if (clmd->clothObject) 
	{
		//clmd->clothObject->old_solver_type = 255;
		clmd->clothObject->edgeset = NULL;
	}
	else 
	{
		printf("Out of memory on allocating clmd->clothObject");
		//BKE_modifier_set_error(ob, &(clmd->modifier), "Out of memory on allocating clmd->clothObject");
		return false;
	}

	/* mesh input objects need Mesh */
	if (!mesh) {
		return false;
	}

	cloth_from_mesh(clmd, ob, mesh);

	/* create springs */
	clmd->clothObject->springs = NULL;
	clmd->clothObject->numsprings = -1;

	clmd->clothObject->sew_edge_graph = NULL;

	//if (clmd->sim_parms->shapekey_rest && !(clmd->sim_parms->flags & CLOTH_SIMSETTINGS_FLAG_DYNAMIC_BASEMESH)) 
	//{
	//	shapekey_rest = CustomData_get_layer(&mesh->vdata, CD_CLOTH_ORCO);
	//}

	mvert = mesh->mvert;

	verts = clmd->clothObject->verts;

	/* set initial values */
	for (i = 0; i < mesh->totvert; i++, verts++) 
	{
		if (first) 
		{
			copy_v3_v3(verts->x, mvert[i].co);

			mul_m4_v3(ob->obmat, verts->x);

			if (shapekey_rest) 
			{
				copy_v3_v3(verts->xrest, shapekey_rest[i]);
				mul_m4_v3(ob->obmat, verts->xrest);
			}
			else 
			{
				copy_v3_v3(verts->xrest, verts->x);
			}
		}

		/* no GUI interface yet */
		verts->mass = clmd->sim_parms->mass;
		verts->impulse_count = 0;

		if (clmd->sim_parms->vgroup_mass > 0) 
		{
			verts->goal = clmd->sim_parms->defgoal;
		}
		else {
			verts->goal = 0.0f;
		}

		verts->shrink_factor = 0.0f;

		verts->flags = 0;
		copy_v3_v3(verts->xold, verts->x);
		copy_v3_v3(verts->xconst, verts->x);
		copy_v3_v3(verts->txold, verts->x);
		copy_v3_v3(verts->tx, verts->x);
		mul_v3_fl(verts->v, 0.0f);

		verts->impulse_count = 0;
		copy_v3_v3(verts->impulse, tnull);
	}

	/* apply / set vertex groups */
	/* has to be happen before springs are build! */
	//cloth_apply_vgroup(clmd, mesh);

	if (!cloth_build_springs(clmd, mesh)) 
	{
		cloth_free_modifier(clmd);
		printf("Cannot build springs");
		//BKE_modifier_set_error(ob, &(clmd->modifier), "Cannot build springs");
		return false;
	}

	/* init our solver */
	SIM_cloth_solver_init(ob, clmd);

	if (!first) 
	{
		SIM_cloth_solver_set_positions(clmd);
	}

	clmd->clothObject->bvhtree = bvhtree_build_from_cloth(clmd, clmd->coll_parms->epsilon);
	clmd->clothObject->bvhselftree = bvhtree_build_from_cloth(clmd, clmd->coll_parms->selfepsilon);

	return true;
}

void BKE_mesh_runtime_looptri_recalc(Mesh* mesh)
{
	mesh_ensure_looptri_data(mesh);
	BLI_assert(mesh->totpoly == 0 || mesh->runtime.looptris.array_wip != nullptr);

	BKE_mesh_recalc_looptri(mesh->mloop, mesh->mpoly,
		mesh->mvert, mesh->totloop, mesh->totpoly,
		mesh->runtime.looptris.array_wip);

	BLI_assert(mesh->runtime.looptris.array == nullptr);
	atomic_cas_ptr((void**)&mesh->runtime.looptris.array, mesh->runtime.looptris.array, mesh->runtime.looptris.array_wip);
	mesh->runtime.looptris.array_wip = nullptr;
}

const MLoopTri* BKE_mesh_runtime_looptri_ensure(const Mesh* mesh)
{
	ThreadMutex* mesh_eval_mutex = (ThreadMutex*)mesh->runtime.eval_mutex;
	BLI_mutex_lock(mesh_eval_mutex);

	MLoopTri* looptri = mesh->runtime.looptris.array;

	if (looptri != nullptr) 
	{
		BLI_assert(BKE_mesh_runtime_looptri_len(mesh) == mesh->runtime.looptris.len);
	}
	else 
	{
		/* Must isolate multithreaded tasks while holding a mutex lock. */
		blender::threading::isolate_task([&]() { BKE_mesh_runtime_looptri_recalc(const_cast<Mesh*>(mesh)); });
		looptri = mesh->runtime.looptris.array;
	}

	BLI_mutex_unlock(mesh_eval_mutex);

	return looptri;
}

void cloth_from_mesh(ClothModifierData* clmd, const Object* ob, Mesh* mesh)
{
	const MLoop* mloop = mesh->mloop;
	const MLoopTri* looptri = BKE_mesh_runtime_looptri_ensure(mesh);
	const unsigned int mvert_num = mesh->totvert;
	const unsigned int looptri_num = mesh->runtime.looptris.len;

	/* Allocate our vertices. */
	clmd->clothObject->mvert_num = mvert_num;
	clmd->clothObject->verts = (ClothVertex*)MEM_callocN(sizeof(ClothVertex) * clmd->clothObject->mvert_num, "clothVertex");
	if (clmd->clothObject->verts == NULL) 
	{
		cloth_free_modifier(clmd);
		//BKE_modifier_set_error(ob, &(clmd->modifier), "Out of memory on allocating clmd->clothObject->verts");
		printf("Out of memory on allocating clmd->clothObject->verts");
		printf("cloth_free_modifier clmd->clothObject->verts\n");
		return;
	}

	/* save face information */
	if (clmd->hairdata == NULL) 
	{
		clmd->clothObject->primitive_num = looptri_num;
	}
	else {
		clmd->clothObject->primitive_num = mesh->totedge;
	}

	clmd->clothObject->tri = (MVertTri*)MEM_mallocN(sizeof(MVertTri) * looptri_num, "clothLoopTris");
	if (clmd->clothObject->tri == NULL) 
	{
		cloth_free_modifier(clmd);
		//BKE_modifier_set_error( ob, &(clmd->modifier), "Out of memory on allocating clmd->clothObject->looptri");
		printf("Out of memory on allocating clmd->clothObject->looptri");
		printf("cloth_free_modifier clmd->clothObject->looptri\n");
		return;
	}
	BKE_mesh_runtime_verttri_from_looptri(clmd->clothObject->tri, mloop, looptri, looptri_num);

	clmd->clothObject->edges = mesh->medge;

	/* Free the springs since they can't be correct if the vertices
	 * changed. */
	if (clmd->clothObject->springs != NULL) 
	{
		MEM_freeN(clmd->clothObject->springs);
	}
}