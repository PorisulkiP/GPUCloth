#include "hip/hip_runtime.h"
#include "MEM_guardedalloc.cuh"

#include "kdopbvh.cuh"
#include "B_math.h"
#include "BLI_stack.h"
#include "mallocn_intern.cuh"
#include "task.cuh"
#include "utildefines.h"

typedef unsigned char axis_t;

enum
{
	MAX_TREETYPE = 32
};

#ifdef DEBUG
#  define KDOPBVH_THREAD_LEAF_THRESHOLD 0
#else
#  define KDOPBVH_THREAD_LEAF_THRESHOLD 1024
#endif

/**
 * Bounding Volume Hierarchy Definition
 *
 * Notes: From OBB until 26-DOP --> all bounding volumes possible, just choose type below
 * Notes: You have to choose the type at compile time ITM
 * Notes: You can choose the tree type --> binary, quad, octree, choose below
 */
__device__ constexpr float d_bvhtree_kdop_axes[13][3] = {
    {1.0, 0, 0},
    {0, 1.0, 0},
    {0, 0, 1.0},
    {1.0, 1.0, 1.0},
    {1.0, -1.0, 1.0},
    {1.0, 1.0, -1.0},
    {1.0, -1.0, -1.0},
    {1.0, 1.0, 0},
    {1.0, 0, 1.0},
    {0, 1.0, 1.0},
    {1.0, -1.0, 0},
    {1.0, 0, -1.0},
    {0, 1.0, -1.0},
};

constexpr float bvhtree_kdop_axes[13][3] = {
    {1.0, 0, 0},
    {0, 1.0, 0},
    {0, 0, 1.0},
    {1.0, 1.0, 1.0},
    {1.0, -1.0, 1.0},
    {1.0, 1.0, -1.0},
    {1.0, -1.0, -1.0},
    {1.0, 1.0, 0},
    {1.0, 0, 1.0},
    {0, 1.0, 1.0},
    {1.0, -1.0, 0},
    {1.0, 0, -1.0},
    {0, 1.0, -1.0},
};

/* Used to correct the epsilon and thus match the overlap distance. */
__device__ constexpr float d_bvhtree_kdop_axes_length[13] = {
    1.0f,
    1.0f,
    1.0f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
};

/* Used to correct the epsilon and thus match the overlap distance. */
static constexpr float bvhtree_kdop_axes_length[13] = {
    1.0f,
    1.0f,
    1.0f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.7320508075688772f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
    1.4142135623730951f,
};

/* -------------------------------------------------------------------- */
/** \name Utility Functions
 * \{ */

__host__ __device__ axis_t min_axis(const axis_t a, const axis_t b)
{
  return (a < b) ? a : b;
}

/**
 * Intro-sort
 * with permission deriving from the following Java code:
 * http://ralphunden.net/content/tutorials/a-guide-to-introsort/
 * and he derived it from the SUN STL
 */

__host__ __device__ static void node_minmax_init(const BVHTree *tree, const BVHNode *node)
{
	const auto bv = reinterpret_cast<float(*)[2]>(node->bv);
#ifdef __CUDA_ARCH__
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < tree->stop_axis && idx > tree->start_axis)
    {
        bv[idx][0] = FLT_MAX;
        bv[idx][1] = -FLT_MAX;
    }
#else
    for (axis_t axis_iter = tree->start_axis; axis_iter != tree->stop_axis; axis_iter++)
    {
        bv[axis_iter][0] = FLT_MAX;
        bv[axis_iter][1] = -FLT_MAX;
    }
#endif
}
/** \} */

/* -------------------------------------------------------------------- */
/** \name Balance Utility Functions
 * \{ */

/**
 * Insertion sort algorithm
 */
static void bvh_insertionsort(BVHNode **a, const int lo, const int hi, const int axis)
{
  int i, j;
  BVHNode *t;
  for (i = lo; i < hi; i++) {
    j = i;
    t = a[i];
    while ((j != lo) && (t->bv[axis] < (a[j - 1])->bv[axis])) {
      a[j] = a[j - 1];
      j--;
    }
    a[j] = t;
  }
}

static int bvh_partition(BVHNode **a, const int lo, const int hi, const BVHNode *x, const int axis)
{
  int i = lo, j = hi;
  while (1) {
    while (a[i]->bv[axis] < x->bv[axis]) {
      i++;
    }
    j--;
    while (x->bv[axis] < a[j]->bv[axis]) {
      j--;
    }
    if (!(i < j)) {
      return i;
    }
    SWAP(BVHNode *, a[i], a[j]);
    i++;
  }
}

/* returns Sortable */
__host__ __device__ static BVHNode *bvh_medianof3(BVHNode **a, const int lo, const int mid, const int hi, const int axis)
{
  if ((a[mid])->bv[axis] < (a[lo])->bv[axis]) {
    if ((a[hi])->bv[axis] < (a[mid])->bv[axis]) {
      return a[mid];
    }
    if ((a[hi])->bv[axis] < (a[lo])->bv[axis]) {
      return a[hi];
    }
    return a[lo];
  }

  if ((a[hi])->bv[axis] < (a[mid])->bv[axis]) {
    if ((a[hi])->bv[axis] < (a[lo])->bv[axis]) {
      return a[lo];
    }
    return a[hi];
  }
  return a[mid];
}

/**
 * \note after a call to this function you can expect one of:
 * - every node to left of a[n] are smaller or equal to it
 * - every node to the right of a[n] are greater or equal to it */
__host__ __device__ static void partition_nth_element(BVHNode** a, int begin, int end, const int n, const int axis)
{
	while (end - begin > 3)
	{
		const int cut = bvh_partition(
			a, begin, end, bvh_medianof3(a, begin, (begin + end) / 2, end - 1, axis), axis);
		if (cut <= n)
		{
			begin = cut;
		}
		else
		{
			end = cut;
		}
	}
	bvh_insertionsort(a, begin, end, axis);
}

/*
 * BVHTree bounding volumes functions
 */
__host__ __device__ static void create_kdop_hull(const BVHTree* tree, const BVHNode* node, const float* co,
                                                 const int numpoints, const int moving)
{
	float* bv = node->bv;

	/* Don't initialize bounds for the moving case */
	if (!moving)
	{
		node_minmax_init(tree, node);
	}

	for (int k = 0; k < numpoints; k++)
	{
		/* for all Axes. */
		for (axis_t axis_iter = tree->start_axis; axis_iter < tree->stop_axis; axis_iter++)
		{
#ifdef __CUDA_ARCH__
    	const float newminmax = dot_v3v3(&co[k * 3], d_bvhtree_kdop_axes[axis_iter]);
#else
			const float newminmax = dot_v3v3(&co[k * 3], bvhtree_kdop_axes[axis_iter]);
#endif
			if (newminmax < bv[2 * axis_iter])
			{
				bv[2 * axis_iter] = newminmax;
			}
			if (newminmax > bv[(2 * axis_iter) + 1])
			{
				bv[(2 * axis_iter) + 1] = newminmax;
			}
		}
	}
}

/**
 * \note depends on the fact that the BVH's for each face is already built
 */
static void refit_kdop_hull(const BVHTree *tree, const BVHNode *node, const int start, const int end)
{
  float newmin, newmax;
  float *__restrict bv = node->bv;
  int j;
  axis_t axis_iter;

  node_minmax_init(tree, node);

  for (j = start; j < end; j++) {
    float *__restrict node_bv = tree->nodes[j]->bv;

    /* for all Axes. */
    for (axis_iter = tree->start_axis; axis_iter < tree->stop_axis; axis_iter++) {
      newmin = node_bv[(2 * axis_iter)];
      if ((newmin < bv[(2 * axis_iter)])) {
        bv[(2 * axis_iter)] = newmin;
      }

      newmax = node_bv[(2 * axis_iter) + 1];
      if ((newmax > bv[(2 * axis_iter) + 1])) {
        bv[(2 * axis_iter) + 1] = newmax;
      }
    }
  }
}

/**
 * only supports x,y,z axis in the moment
 * but we should use a plain and simple function here for speed sake */
__host__ __device__ char get_largest_axis(const float* bv)
{
#ifdef __CUDA_ARCH__
    // GPU-specific code
    float middle_point[3];

    middle_point[0] = __fsub_rn(bv[1], bv[0]); // x axis
    middle_point[1] = __fsub_rn(bv[3], bv[2]); // y axis
    middle_point[2] = __fsub_rn(bv[5], bv[4]); // z axis

    if (__fadd_rn(middle_point[0], middle_point[1]) > 0)
    {
        if (__fadd_rn(middle_point[0], middle_point[2]) > 0)
        {
            return 1; // max x axis
        }
        return 5; // max z axis
    }
    if (__fadd_rn(middle_point[1], middle_point[2]) > 0)
    {
        return 3; // max y axis
    }
    return 5; // max z axis

#else
    // CPU-specific code
    float middle_point[3];

    middle_point[0] = (bv[1]) - (bv[0]); // x axis
    middle_point[1] = (bv[3]) - (bv[2]); // y axis
    middle_point[2] = (bv[5]) - (bv[4]); // z axis

    if (middle_point[0] > middle_point[1])
    {
        if (middle_point[0] > middle_point[2])
        {
            return 1; // max x axis
        }
        return 5; // max z axis
    }
    if (middle_point[1] > middle_point[2])
    {
        return 3; // max y axis
    }
    return 5; // max z axis
#endif
}


/**
 * bottom-up update of bvh node BV
 * join the children on the parent BV */
__host__ __device__ void node_join(const BVHTree* tree, const BVHNode* node)
{
#ifdef __CUDA_ARCH__
    // GPU-specific code
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    axis_t axis_iter = tree->start_axis + idx;

    if (idx < tree->tree_type && node->children[idx]) 
    {
        if (axis_iter >= tree->start_axis && axis_iter < tree->stop_axis) 
        {
            // update minimum
            if (node->children[idx]->bv[(2 * axis_iter)] < node->bv[(2 * axis_iter)]) 
            {
                node->bv[(2 * axis_iter)] = node->children[idx]->bv[(2 * axis_iter)];
            }
            // update maximum
            if (node->children[idx]->bv[(2 * axis_iter) + 1] > node->bv[(2 * axis_iter) + 1]) 
            {
                node->bv[(2 * axis_iter) + 1] = node->children[idx]->bv[(2 * axis_iter) + 1];
            }
        }
    }

#else
    // CPU-specific code

    for (int i = 0; i < tree->tree_type; i++) 
    {
        if (node->children[i]) 
        {
            for (axis_t axis_iter = tree->start_axis; axis_iter < tree->stop_axis; axis_iter++) 
            {
                // update minimum
                if (node->children[i]->bv[(2 * axis_iter)] < node->bv[(2 * axis_iter)]) {
                    node->bv[(2 * axis_iter)] = node->children[i]->bv[(2 * axis_iter)];
                }
                // update maximum
                if (node->children[i]->bv[(2 * axis_iter) + 1] > node->bv[(2 * axis_iter) + 1]) 
                {
                    node->bv[(2 * axis_iter) + 1] = node->children[i]->bv[(2 * axis_iter) + 1];
                }
            }
        }
        else {
            break;
        }
    }
#endif
}


#ifdef USE_PRINT_TREE

/**
 * Debug and information functions
 */

static void bvhtree_print_tree(BVHTree *tree, BVHNode *node, int depth)
{
  int i;
  axis_t axis_iter;

  for (i = 0; i < depth; i++) {
    printf(" ");
  }
  printf(" - %d (%ld): ", node->index, (long int)(node - tree->nodearray));
  for (axis_iter = (axis_t)(2 * tree->start_axis); axis_iter < (axis_t)(2 * tree->stop_axis);
       axis_iter++) {
    printf("%.3f ", node->bv[axis_iter]);
  }
  printf("\n");

  for (i = 0; i < tree->tree_type; i++) {
    if (node->children[i]) {
      bvhtree_print_tree(tree, node->children[i], depth + 1);
    }
  }
}

static void bvhtree_info(BVHTree *tree)
{
  printf("BVHTree Info: tree_type = %d, axis = %d, epsilon = %f\n",
         tree->tree_type,
         tree->axis,
         tree->epsilon);
  printf("nodes = %d, branches = %d, leafs = %d\n",
         tree->totbranch + tree->totleaf,
         tree->totbranch,
         tree->totleaf);
  printf(
      "Memory per node = %ubytes\n",
      (uint)(sizeof(BVHNode) + sizeof(BVHNode *) * tree->tree_type + sizeof(float) * tree->axis));
  printf("BV memory = %ubytes\n", (uint)MEM_lockfree_allocN_len(tree->nodebv));

  printf("Total memory = %ubytes\n",
         (uint)(sizeof(BVHTree) + MEM_lockfree_allocN_len(tree->nodes) + MEM_lockfree_allocN_len(tree->nodearray) +
                MEM_lockfree_allocN_len(tree->nodechild) + MEM_lockfree_allocN_len(tree->nodebv)));

  bvhtree_print_tree(tree, tree->nodes[tree->totleaf], 0);
}
#endif /* USE_PRINT_TREE */

#ifdef USE_VERIFY_TREE

static void bvhtree_verify(BVHTree *tree)
{
  int i, j, check = 0;

  /* check the pointer list */
  for (i = 0; i < tree->totleaf; i++) {
    if (tree->nodes[i]->parent == NULL) {
      printf("Leaf has no parent: %d\n", i);
    }
    else {
      for (j = 0; j < tree->tree_type; j++) {
        if (tree->nodes[i]->parent->children[j] == tree->nodes[i]) {
          check = 1;
        }
      }
      if (!check) {
        printf("Parent child relationship doesn't match: %d\n", i);
      }
      check = 0;
    }
  }

  /* check the leaf list */
  for (i = 0; i < tree->totleaf; i++) {
    if (tree->nodearray[i].parent == NULL) {
      printf("Leaf has no parent: %d\n", i);
    }
    else {
      for (j = 0; j < tree->tree_type; j++) {
        if (tree->nodearray[i].parent->children[j] == &tree->nodearray[i]) {
          check = 1;
        }
      }
      if (!check) {
        printf("Parent child relationship doesn't match: %d\n", i);
      }
      check = 0;
    }
  }

  printf("branches: %d, leafs: %d, total: %d\n",
         tree->totbranch,
         tree->totleaf,
         tree->totbranch + tree->totleaf);
}
#endif /* USE_VERIFY_TREE */

/* Helper data and structures to build a min-leaf generalized implicit tree
 * This code can be easily reduced
 * (basically this is only method to calculate pow(k, n) in O(1).. and stuff like that) */
typedef struct BVHBuildHelper {
  int tree_type;
  int totleafs;

  /** Min number of leafs that are achievable from a node at depth `N`. */
  int leafs_per_child[32];
  /** Number of nodes at depth `N (tree_type^N)`. */
  int branches_on_level[32];

  /** Number of leafs that are placed on the level that is not 100% filled */
  int remain_leafs;

} BVHBuildHelper;

__host__ __device__ void build_implicit_tree_helper(const BVHTree *tree, BVHBuildHelper *data)
{
  int depth = 0;

  data->totleafs = tree->totleaf;
  data->tree_type = tree->tree_type;

  /* Calculate the smallest tree_type^n such that tree_type^n >= num_leafs */
  for (data->leafs_per_child[0] = 1; data->leafs_per_child[0] < data->totleafs;
       data->leafs_per_child[0] *= data->tree_type) {
    /* pass */
  }

  data->branches_on_level[0] = 1;

  for (depth = 1; (depth < 32) && data->leafs_per_child[depth - 1]; depth++) {
    data->branches_on_level[depth] = data->branches_on_level[depth - 1] * data->tree_type;
    data->leafs_per_child[depth] = data->leafs_per_child[depth - 1] / data->tree_type;
  }

  const int remain = data->totleafs - data->leafs_per_child[1];
  const int nnodes = (remain + data->tree_type - 2) / (data->tree_type - 1);
  data->remain_leafs = remain + nnodes;
}

/**
 * Return the min index of all the leafs achievable with the given branch.
 */
static int implicit_leafs_index(const BVHBuildHelper *data, const int depth, const int child_index)
{
  int min_leaf_index = child_index * data->leafs_per_child[depth - 1];
  if (min_leaf_index <= data->remain_leafs) {
    return min_leaf_index;
  }
  if (data->leafs_per_child[depth]) {
    return data->totleafs -
           (data->branches_on_level[depth - 1] - child_index) * data->leafs_per_child[depth];
  }
  return data->remain_leafs;
}

/**
 * Generalized implicit tree build
 *
 * An implicit tree is a tree where its structure is implied,
 * thus there is no need to store child pointers or indexes.
 * It's possible to find the position of the child or the parent with simple maths
 * (multiplication and addition).
 * This type of tree is for example used on heaps..
 * where node N has its child at indices N*2 and N*2+1.
 *
 * Although in this case the tree type is general.. and not know until run-time.
 * tree_type stands for the maximum number of children that a tree node can have.
 * All tree types >= 2 are supported.
 *
 * Advantages of the used trees include:
 * - No need to store child/parent relations (they are implicit);
 * - Any node child always has an index greater than the parent;
 * - Brother nodes are sequential in memory;
 * Some math relations derived for general implicit trees:
 *
 *   K = tree_type, ( 2 <= K )
 *   ROOT = 1
 *   N child of node A = A * K + (2 - K) + N, (0 <= N < K)
 *
 * Util methods:
 *   TODO...
 *    (looping elements, knowing if its a leaf or not.. etc...)
 */

/* This functions returns the number of branches needed to have the requested number of leafs. */
__host__ __device__ static int implicit_needed_branches(const int tree_type, const int leafs)
{
  return max_ii(1, (leafs + tree_type - 3) / (tree_type - 1));
}

/**
 * This function handles the problem of "sorting" the leafs (along the split_axis).
 *
 * It arranges the elements in the given partitions such that:
 * - any element in partition N is less or equal to any element in partition N+1.
 * - if all elements are different all partition will get the same subset of elements
 *   as if the array was sorted.
 *
 * partition P is described as the elements in the range ( nth[P], nth[P+1] ]
 *
 * TODO: This can be optimized a bit by doing a specialized nth_element instead of K nth_elements
 */
__host__ __device__ static void split_leafs(BVHNode **leafs_array,
                        const int nth[],
                        const int partitions,
                        const int split_axis)
{
  int i;
  for (i = 0; i < partitions - 1; i++) {
    if (nth[i] >= nth[partitions]) {
      break;
    }

    partition_nth_element(leafs_array, nth[i], nth[partitions], nth[i + 1], split_axis);
  }
}

typedef struct BVHDivNodesData {
  const BVHTree *tree;
  BVHNode *branches_array;
  BVHNode **leafs_array;

  int tree_type;
  int tree_offset;

  const BVHBuildHelper *data;

  int depth;
  int i;
  int first_of_next_level;
} BVHDivNodesData;

static void non_recursive_bvh_div_nodes_task_cb(void *__restrict userdata, const int j)
{
  BVHDivNodesData *data = (BVHDivNodesData*)userdata;

  int k;
  const int parent_level_index = j - data->i;
  BVHNode *parent = &data->branches_array[j];
  int nth_positions[MAX_TREETYPE + 1];
  char split_axis;

  int parent_leafs_begin = implicit_leafs_index(data->data, data->depth, parent_level_index);
  int parent_leafs_end = implicit_leafs_index(data->data, data->depth, parent_level_index + 1);

  /* This calculates the bounding box of this branch
   * and chooses the largest axis as the axis to divide leafs */
  refit_kdop_hull(data->tree, parent, parent_leafs_begin, parent_leafs_end);
  split_axis = get_largest_axis(parent->bv);

  /* Save split axis (this can be used on ray-tracing to speedup the query time) */
  parent->main_axis = split_axis / 2;

  /* Split the children along the split_axis, note: its not needed to sort the whole leafs array
   * Only to assure that the elements are partitioned on a way that each child takes the elements
   * it would take in case the whole array was sorted.
   * Split_leafs takes care of that "sort" problem. */
  nth_positions[0] = parent_leafs_begin;
  nth_positions[data->tree_type] = parent_leafs_end;
  for (k = 1; k < data->tree_type; k++) {
    const int child_index = j * data->tree_type + data->tree_offset + k;
    /* child level index */
    const int child_level_index = child_index - data->first_of_next_level;
    nth_positions[k] = implicit_leafs_index(data->data, data->depth + 1, child_level_index);
  }

  split_leafs(data->leafs_array, nth_positions, data->tree_type, split_axis);

  /* Setup children and totnode counters
   * Not really needed but currently most of BVH code
   * relies on having an explicit children structure */
  for (k = 0; k < data->tree_type; k++) {
    const int child_index = j * data->tree_type + data->tree_offset + k;
    /* child level index */
    const int child_level_index = child_index - data->first_of_next_level;

    const int child_leafs_begin = implicit_leafs_index(
        data->data, data->depth + 1, child_level_index);
    const int child_leafs_end = implicit_leafs_index(
        data->data, data->depth + 1, child_level_index + 1);

    if (child_leafs_end - child_leafs_begin > 1) {
      parent->children[k] = &data->branches_array[child_index];
      parent->children[k]->parent = parent;
    }
    else if (child_leafs_end - child_leafs_begin == 1) {
      parent->children[k] = data->leafs_array[child_leafs_begin];
      parent->children[k]->parent = parent;
    }
    else {
      break;
    }
  }
  parent->totnode = (char)k;
}

/**
 * This functions builds an optimal implicit tree from the given leafs.
 * Where optimal stands for:
 * - The resulting tree will have the smallest number of branches;
 * - At most only one branch will have NULL children;
 * - All leafs will be stored at level N or N+1.
 *
 * This function creates an implicit tree on branches_array,
 * the leafs are given on the leafs_array.
 *
 * The tree is built per depth levels. First branches at depth 1.. then branches at depth 2.. etc..
 * The reason is that we can build level N+1 from level N without any data dependencies..
 * thus it allows to use multi-thread building.
 *
 * To archive this is necessary to find how much leafs are accessible from a certain branch,
 * #BVHBuildHelper, #implicit_needed_branches and #implicit_leafs_index
 * are auxiliary functions to solve that "optimal-split".
 */
__host__ __device__ void non_recursive_bvh_div_nodes(const BVHTree *tree, BVHNode *branches_array, BVHNode **leafs_array, const int num_leafs)
{
  int i;

  const int tree_type = tree->tree_type;
  /* this value is 0 (on binary trees) and negative on the others */
  const int tree_offset = 2 - tree->tree_type;

  const int num_branches = implicit_needed_branches(tree_type, num_leafs);

  BVHBuildHelper data;
  int depth;

  /* set parent from root node to NULL */
  BVHNode* root = &branches_array[1];
  root->parent = nullptr;

  /* Most of bvhtree code relies on 1-leaf trees having at least one branch
   * We handle that special case here */
  if (num_leafs == 1) 
  {
      refit_kdop_hull(tree, root, 0, num_leafs);
      root->main_axis = get_largest_axis(root->bv) / 2;
      root->totnode = 1;
      root->children[0] = leafs_array[0];
      root->children[0]->parent = root;
      return;
  }

  build_implicit_tree_helper(tree, &data);

  BVHDivNodesData cb_data = {
      .tree = tree,
      .branches_array = branches_array,
      .leafs_array = leafs_array,
      .tree_type = tree_type,
      .tree_offset = tree_offset,
      .data = &data,
      .first_of_next_level = 0
  };

  /* Loop tree levels (log N) loops */
  for (i = 1, depth = 1; i <= num_branches; i = i * tree_type + tree_offset, depth++) 
  {
    const int first_of_next_level = i * tree_type + tree_offset;
    /* index of last branch on this level */
    const int i_stop = min_ii(first_of_next_level, num_branches + 1);

    /* Loop all branches on this level */
    cb_data.first_of_next_level = first_of_next_level;
    cb_data.i = i;
    cb_data.depth = depth;

    if constexpr (true)
    {
      TaskParallelSettings settings;
      BLI_parallel_range_settings_defaults(&settings);
      settings.use_threading = (num_leafs > KDOPBVH_THREAD_LEAF_THRESHOLD);
      BLI_task_parallel_range(i, i_stop, &cb_data, reinterpret_cast<TaskParallelRangeFunc>(non_recursive_bvh_div_nodes_task_cb), &settings);
    }
    else 
    {
      /* Less hassle for debugging. */
      TaskParallelTLS tls = {nullptr};
      for (int i_task = i; i_task < i_stop; i_task++) {
        non_recursive_bvh_div_nodes_task_cb(&cb_data, i_task);
      }
    }
  }
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree API
 * \{ */

/**
 * \note many callers don't check for ``NULL`` return.
 */
__host__ __device__ BVHTree *BLI_bvhtree_new(const uint maxsize, float epsilon, const char tree_type, const char axis)
{
  BVHTree *tree;
  int numnodes, i;

  BLI_assert(tree_type >= 2 && tree_type <= MAX_TREETYPE);

  tree = (BVHTree*)MEM_lockfree_callocN(sizeof(BVHTree), "BVHTree");

  /* tree epsilon must be >= FLT_EPSILON
   * so that tangent rays can still hit a bounding volume..
   * this bug would show up when casting a ray aligned with a kdop-axis
   * and with an edge of 2 faces */
  epsilon = max_ff(FLT_EPSILON, epsilon);

  if (tree) {
    tree->epsilon = epsilon;
    tree->tree_type = tree_type;
    tree->axis = axis;

    if (axis == 26) {
      tree->start_axis = 0;
      tree->stop_axis = 13;
    }
    else if (axis == 18) {
      tree->start_axis = 7;
      tree->stop_axis = 13;
    }
    else if (axis == 14) {
      tree->start_axis = 0;
      tree->stop_axis = 7;
    }
    else if (axis == 8) { /* AABB */
      tree->start_axis = 0;
      tree->stop_axis = 4;
    }
    else if (axis == 6) { /* OBB */
      tree->start_axis = 0;
      tree->stop_axis = 3;
    }
    else {
      /* should never happen! */
      BLI_assert(0);

      goto fail;
    }

    /* Allocate arrays */
    numnodes = maxsize + implicit_needed_branches(tree_type, maxsize) + tree_type;

    tree->nodes = (BVHNode**)MEM_lockfree_callocN(sizeof(BVHNode *) * (size_t)numnodes, "BVHNodes");
    tree->nodebv = (float*)MEM_lockfree_callocN(sizeof(float) * (size_t)(axis * numnodes), "BVHNodeBV");
    tree->nodechild = (BVHNode**)MEM_lockfree_callocN(sizeof(BVHNode *) * (size_t)(tree_type * numnodes), "BVHNodeBV");
    tree->nodearray = (BVHNode*)MEM_lockfree_callocN(sizeof(BVHNode) * (size_t)numnodes, "BVHNodeArray");

    if (UNLIKELY((!tree->nodes) || (!tree->nodebv) || (!tree->nodechild) || (!tree->nodearray))) {
      goto fail;
    }

    /* link the dynamic bv and child links */
    for (i = 0; i < numnodes; i++) {
      tree->nodearray[i].bv = &tree->nodebv[i * axis];
      tree->nodearray[i].children = &tree->nodechild[i * tree_type];
    }
  }
  return tree;

fail:
  BLI_bvhtree_free(tree);
  return nullptr;
}

__host__ __device__ void BLI_bvhtree_free(BVHTree *tree)
{
  if (tree) {
    MEM_SAFE_FREE(tree->nodes);
    MEM_SAFE_FREE(tree->nodearray);
    MEM_SAFE_FREE(tree->nodebv);
    MEM_SAFE_FREE(tree->nodechild);
    MEM_lockfree_freeN(tree);
  }
}

__host__ __device__ void BLI_bvhtree_balance(BVHTree *tree)
{
  BVHNode **leafs_array = tree->nodes;

  /* This function should only be called once
   * (some big bug goes here if its being called more than once per tree) */
  BLI_assert(tree->totbranch == 0);

  /* Build the implicit tree */
  non_recursive_bvh_div_nodes(tree, tree->nodearray + (tree->totleaf - 1), leafs_array, tree->totleaf);

  /* current code expects the branches to be linked to the nodes array
   * we perform that linkage here */
  tree->totbranch = implicit_needed_branches(tree->tree_type, tree->totleaf);

#ifdef __CUDA_ARCH__
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < tree->totbranch)
  {
    tree->nodes[tree->totleaf + idx] = &tree->nodearray[tree->totleaf + idx];
  }
#else
  for (int i = 0; i < tree->totbranch; i++)
  {
      tree->nodes[tree->totleaf + i] = &tree->nodearray[tree->totleaf + i];
  }
#endif	

#ifdef USE_SKIP_LINKS
  build_skip_links(tree, tree->nodes[tree->totleaf], NULL, NULL);
#endif

#ifdef USE_VERIFY_TREE
  bvhtree_verify(tree);
#endif

#ifdef USE_PRINT_TREE
  bvhtree_info(tree);
#endif
}

__host__ __device__ void bvhtree_node_inflate(const BVHTree* tree, const BVHNode* node, const float dist)
{
#ifdef __CUDA_ARCH__
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tree->start_axis && idx < tree->stop_axis)
    {
        float dist_corrected = dist * d_bvhtree_kdop_axes_length[idx];
        node->bv[(2 * idx)] -= dist_corrected; /* minimum */
        node->bv[(2 * idx) + 1] += dist_corrected; /* maximum */
    }
#else
    for (axis_t axis_iter = tree->start_axis; axis_iter < tree->stop_axis; axis_iter++)
    {
		float dist_corrected = dist * bvhtree_kdop_axes_length[axis_iter];
        node->bv[(2 * axis_iter)] -= dist_corrected; /* minimum */
        node->bv[(2 * axis_iter) + 1] += dist_corrected; /* maximum */	
    }
#endif	
}

__host__ __device__ void BLI_bvhtree_insert(BVHTree* tree, const int index, const float co[3], const int numpoints)
{
	/* insert should only possible as long as tree->totbranch is 0 */
	BLI_assert(tree->totbranch <= 0);
	BLI_assert((size_t)tree->totleaf < MEM_lockfree_allocN_len(tree->nodes) / sizeof(*(tree->nodes)));

	BVHNode* node = tree->nodes[tree->totleaf] = &(tree->nodearray[tree->totleaf]);
	tree->totleaf++;

	create_kdop_hull(tree, node, co, numpoints, 0);
	node->index = index;

	/* inflate the bv with some epsilon */
	bvhtree_node_inflate(tree, node, tree->epsilon);
}

/* call before BLI_bvhtree_update_tree() */
__host__ __device__ bool BLI_bvhtree_update_node(const BVHTree *tree, const int index, const float co[3], const float co_moving[3], const int numpoints)
{
	/* check if index exists */
  if (index > tree->totleaf) {
    return false;
  }

  BVHNode* node = tree->nodearray + index;

  create_kdop_hull(tree, node, co, numpoints, 0);

  if (co_moving) {
    create_kdop_hull(tree, node, co_moving, numpoints, 1);
  }

  /* inflate the bv with some epsilon */
  bvhtree_node_inflate(tree, node, tree->epsilon);

  return true;
}
/**
 * Call #BLI_bvhtree_update_node() first for every node/point/triangle.
 */
__host__ __device__ void BLI_bvhtree_update_tree(BVHTree *tree)
{
  /* Update bottom=>top
   * TRICKY: the way we build the tree all the children have an index greater than the parent
   * This allows us todo a bottom up update by starting on the bigger numbered branch. */

  BVHNode **root = tree->nodes + tree->totleaf;
  BVHNode **index = tree->nodes + tree->totleaf + tree->totbranch - 1;

  for (; index >= root; index--) {
    node_join(tree, *index);
  }
}

/**
 * Number of times #BLI_bvhtree_insert has been called.
 * mainly useful for asserts functions to check we added the correct number.
 */
__host__ __device__ int BLI_bvhtree_get_len(const BVHTree *tree)
{
  return tree->totleaf;
}

/**
 * Maximum number of children that a node can have.
 */
__host__ __device__ int BLI_bvhtree_get_tree_type(const BVHTree *tree)
{
  return tree->tree_type;
}

__host__ __device__ float BLI_bvhtree_get_epsilon(const BVHTree *tree)
{
  return tree->epsilon;
}

/**
 * This function returns the bounding box of the BVH tree.
 */
void BLI_bvhtree_get_bounding_box(const BVHTree *tree, float r_bb_min[3], float r_bb_max[3])
{
  BVHNode *root = tree->nodes[tree->totleaf];
  if (root != nullptr) {
    const float bb_min[3] = {root->bv[0], root->bv[2], root->bv[4]};
    const float bb_max[3] = {root->bv[1], root->bv[3], root->bv[5]};
    copy_v3_v3(r_bb_min, bb_min);
    copy_v3_v3(r_bb_max, bb_max);
  }
  else {
    BLI_assert(false);
    zero_v3(r_bb_min);
    zero_v3(r_bb_max);
  }
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_overlap
 * \{ */

/**
 * overlap - is it possible for 2 bv's to collide ?
 */
static bool tree_overlap_test(const BVHNode *node1,
                              const BVHNode *node2,
                              const axis_t start_axis,
                              const axis_t stop_axis)
{
  const float *bv1 = node1->bv + (start_axis << 1);
  const float *bv2 = node2->bv + (start_axis << 1);
  const float *bv1_end = node1->bv + (stop_axis << 1);

  /* test all axis if min + max overlap */
  for (; bv1 != bv1_end; bv1 += 2, bv2 += 2) {
    if ((bv1[0] > bv2[1]) || (bv2[0] > bv1[1])) {
      return 0;
    }
  }

  return 1;
}

__device__ static bool d_tree_overlap_test(const BVHNode* node1,
    const BVHNode* node2,
    const axis_t start_axis,
    const axis_t stop_axis)
{
    const float* bv1 = node1->bv + (start_axis << 1);
    const float* bv2 = node2->bv + (start_axis << 1);
    const float* bv1_end = node1->bv + (stop_axis << 1);

    /* test all axis if min + max overlap */
    for (; bv1 != bv1_end; bv1 += 2, bv2 += 2) {
        if ((bv1[0] > bv2[1]) || (bv2[0] > bv1[1])) {
            return false;
        }
    }

    return true;
}

static void tree_overlap_traverse(BVHOverlapData_Thread *data_thread,
                                  const BVHNode *node1,
                                  const BVHNode *node2)
{
  BVHOverlapData_Shared *data = (BVHOverlapData_Shared*)data_thread->shared;
  int j;

  if (tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) {
    /* check if node1 is a leaf */
    if (!node1->totnode) {
      /* check if node2 is a leaf */
      if (!node2->totnode) {
        BVHTreeOverlap *overlap;

        if (UNLIKELY(node1 == node2)) {
          return;
        }

        /* both leafs, insert overlap! */
        overlap = (BVHTreeOverlap*)BLI_stack_push_r(data_thread->overlap);
        overlap->indexA = node1->index;
        overlap->indexB = node2->index;
      }
      else {
        for (j = 0; j < data->tree2->tree_type; ++j) {
          if (node2->children[j]) {
            tree_overlap_traverse(data_thread, node1, node2->children[j]);
          }
        }
      }
    }
    else {
      for (j = 0; j < data->tree1->tree_type; ++j) {
        if (node1->children[j]) {
          tree_overlap_traverse(data_thread, node1->children[j], node2);
        }
      }
    }
  }
}

__device__ static void d_tree_overlap_traverse(BVHOverlapData_Thread* data_thread,
    const BVHNode* node1,
    const BVHNode* node2)
{
	const auto* data = data_thread->shared;
    int j;

    if (d_tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) {
        /* check if node1 is a leaf */
        if (!node1->totnode) {
            /* check if node2 is a leaf */
            if (!node2->totnode) {
	            if (UNLIKELY(node1 == node2)) {
                    return;
                }

                /* both leafs, insert overlap! */
	            auto overlap = static_cast<BVHTreeOverlap*>(BLI_stack_push_r(data_thread->overlap));
                overlap->indexA = node1->index;
                overlap->indexB = node2->index;
            }
            else {
                for (j = 0; j < data->tree2->tree_type; ++j) {
                    if (node2->children[j]) {
                        d_tree_overlap_traverse(data_thread, node1, node2->children[j]);
                    }
                }
            }
        }
        else {
            for (j = 0; j < data->tree1->tree_type; ++j) {
                if (node1->children[j]) {
                    d_tree_overlap_traverse(data_thread, node1->children[j], node2);
                }
            }
        }
    }
}

/**
 * a version of #tree_overlap_traverse that runs a callback to check if the nodes really intersect.
 */
static void tree_overlap_traverse_cb(BVHOverlapData_Thread *data_thread,
                                     const BVHNode *node1,
                                     const BVHNode *node2)
{
  BVHOverlapData_Shared *data = data_thread->shared;
  int j;

  if (tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) {
    /* check if node1 is a leaf */
    if (!node1->totnode) {
      /* check if node2 is a leaf */
      if (!node2->totnode) {
        BVHTreeOverlap *overlap;

        if (UNLIKELY(node1 == node2)) {
          return;
        }

        /* only difference to tree_overlap_traverse! */
        if (data->callback(data->userdata, node1->index, node2->index, data_thread->thread)) {
          /* both leafs, insert overlap! */
          overlap = (BVHTreeOverlap*)BLI_stack_push_r(data_thread->overlap);
          overlap->indexA = node1->index;
          overlap->indexB = node2->index;
        }
      }
      else {
        for (j = 0; j < data->tree2->tree_type; j++) {
          if (node2->children[j]) {
            tree_overlap_traverse_cb(data_thread, node1, node2->children[j]);
          }
        }
      }
    }
    else {
      for (j = 0; j < data->tree1->tree_type; j++) {
        if (node1->children[j]) {
          tree_overlap_traverse_cb(data_thread, node1->children[j], node2);
        }
      }
    }
  }
}

__device__ static void d_tree_overlap_traverse_cb(BVHOverlapData_Thread* data_thread,
    const BVHNode* node1,
    const BVHNode* node2)
{
	const BVHOverlapData_Shared* data = data_thread->shared;
    int j;

    if (d_tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) 
    {
        /* check if node1 is a leaf */
        if (!node1->totnode) 
        {
            /* check if node2 is a leaf */
            if (!node2->totnode) 
            {
	            if (UNLIKELY(node1 == node2)) 
                {
                    return;
                }

                /* only difference to tree_overlap_traverse! */
                if (data->callback(data->userdata, node1->index, node2->index, data_thread->thread)) 
                {
                    /* both leafs, insert overlap! */
                    auto* overlap = static_cast<BVHTreeOverlap*>(BLI_stack_push_r(data_thread->overlap));
                    overlap->indexA = node1->index;
                    overlap->indexB = node2->index;
                }
            }
            else
            {
                for (j = 0; j < data->tree2->tree_type; j++) 
                {
                    if (node2->children[j]) {
                        d_tree_overlap_traverse_cb(data_thread, node1, node2->children[j]);
                    }
                }
            }
        }
        else 
        {
            for (j = 0; j < data->tree1->tree_type; j++) 
            {
                if (node1->children[j]) 
                {
                    d_tree_overlap_traverse_cb(data_thread, node1->children[j], node2);
                }
            }
        }
    }
}


/**
 * a version of #tree_overlap_traverse_cb that that break on first true return.
 */
static bool tree_overlap_traverse_num(BVHOverlapData_Thread *data_thread,
                                      const BVHNode *node1,
                                      const BVHNode *node2)
{
  BVHOverlapData_Shared *data = data_thread->shared;
  int j;

  if (tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) {
    /* check if node1 is a leaf */
    if (!node1->totnode) {
      /* check if node2 is a leaf */
      if (!node2->totnode) {
        BVHTreeOverlap *overlap;

        if (UNLIKELY(node1 == node2)) {
          return false;
        }

        /* only difference to tree_overlap_traverse! */
        if (!data->callback ||
            data->callback(data->userdata, node1->index, node2->index, data_thread->thread)) {
          /* both leafs, insert overlap! */
          if (data_thread->overlap) {
            overlap = (BVHTreeOverlap*)BLI_stack_push_r(data_thread->overlap);
            overlap->indexA = node1->index;
            overlap->indexB = node2->index;
          }
          return (--data_thread->max_interactions) == 0;
        }
      }
      else {
        for (j = 0; j < node2->totnode; j++) {
          if (tree_overlap_traverse_num(data_thread, node1, node2->children[j])) {
            return true;
          }
        }
      }
    }
    else {
      const uint max_interactions = data_thread->max_interactions;
      for (j = 0; j < node1->totnode; j++) {
        if (tree_overlap_traverse_num(data_thread, node1->children[j], node2)) {
          data_thread->max_interactions = max_interactions;
        }
      }
    }
  }
  return false;
}

__device__ static bool d_tree_overlap_traverse_num(BVHOverlapData_Thread* data_thread,
    const BVHNode* node1,
    const BVHNode* node2)
{
	const BVHOverlapData_Shared* data = data_thread->shared;
    int j;

    if (d_tree_overlap_test(node1, node2, data->start_axis, data->stop_axis)) {
        /* check if node1 is a leaf */
        if (!node1->totnode) {
            /* check if node2 is a leaf */
            if (!node2->totnode) {
	            if (UNLIKELY(node1 == node2)) {
                    return false;
                }

                /* only difference to tree_overlap_traverse! */
                if (!data->callback ||
                    data->callback(data->userdata, node1->index, node2->index, data_thread->thread)) {
                    /* both leafs, insert overlap! */
                    if (data_thread->overlap) {
	                    auto* overlap = static_cast<BVHTreeOverlap*>(BLI_stack_push_r(data_thread->overlap));
                        overlap->indexA = node1->index;
                        overlap->indexB = node2->index;
                    }
                    return (--data_thread->max_interactions) == 0;
                }
            }
            else {
                for (j = 0; j < node2->totnode; j++) {
                    if (d_tree_overlap_traverse_num(data_thread, node1, node2->children[j])) {
                        return true;
                    }
                }
            }
        }
        else {
            const uint max_interactions = data_thread->max_interactions;
            for (j = 0; j < node1->totnode; j++) {
                if (d_tree_overlap_traverse_num(data_thread, node1->children[j], node2)) {
                    data_thread->max_interactions = max_interactions;
                }
            }
        }
    }
    return false;
}

/**
 * Use to check the total number of threads #BLI_bvhtree_overlap will use.
 *
 * \warning Must be the first tree passed to #BLI_bvhtree_overlap!
 */
__host__ __device__ int BLI_bvhtree_overlap_thread_num(const BVHTree *tree)
{
  return (int)MIN2(tree->tree_type, tree->nodes[tree->totleaf]->totnode);
}

__host__ __device__ static void bvhtree_overlap_task_cb(void *__restrict userdata, const int j)
{
  BVHOverlapData_Thread *data = &((BVHOverlapData_Thread *)userdata)[j];
  BVHOverlapData_Shared *data_shared = data->shared;

  if (data->max_interactions) {
    tree_overlap_traverse_num(data,
                              data_shared->tree1->nodes[data_shared->tree1->totleaf]->children[j],
                              data_shared->tree2->nodes[data_shared->tree2->totleaf]);
  }
  else if (data_shared->callback) {
    tree_overlap_traverse_cb(data,
                             data_shared->tree1->nodes[data_shared->tree1->totleaf]->children[j],
                             data_shared->tree2->nodes[data_shared->tree2->totleaf]);
  }
  else {
    tree_overlap_traverse(data,
                          data_shared->tree1->nodes[data_shared->tree1->totleaf]->children[j],
                          data_shared->tree2->nodes[data_shared->tree2->totleaf]);
  }
}

__host__ __device__ BVHTreeOverlap *BLI_bvhtree_overlap_ex(
    const BVHTree *tree1,
    const BVHTree *tree2,
    uint *r_overlap_tot,
    /* optional callback to test the overlap before adding (must be thread-safe!) */
    const BVHTree_OverlapCallback callback,
    void *userdata,
    const uint max_interactions,
    const int flag)
{
  bool overlap_pairs = (flag & BVH_OVERLAP_RETURN_PAIRS) != 0;
  bool use_threading = (flag & BVH_OVERLAP_USE_THREADING) != 0 &&
                       (tree1->totleaf > KDOPBVH_THREAD_LEAF_THRESHOLD);

  /* 'RETURN_PAIRS' was not implemented without 'max_interactions'. */
  BLI_assert(overlap_pairs || max_interactions);

  const int root_node_len = BLI_bvhtree_overlap_thread_num(tree1);
  const int thread_num = use_threading ? root_node_len : 1;
  int j;
  size_t total = 0;
  BVHTreeOverlap *overlap = nullptr, *to = nullptr;
  BVHOverlapData_Shared data_shared;
  BVHOverlapData_Thread *data = static_cast<BVHOverlapData_Thread*>(BLI_array_alloca(data, static_cast<size_t>(thread_num)));
  axis_t start_axis, stop_axis;

  /* check for compatibility of both trees (can't compare 14-DOP with 18-DOP) */
  if (UNLIKELY((tree1->axis != tree2->axis) && (tree1->axis == 14 || tree2->axis == 14) &&
               (tree1->axis == 18 || tree2->axis == 18))) {
    BLI_assert(0);
    return nullptr;
  }

  const BVHNode *root1 = tree1->nodes[tree1->totleaf];
  const BVHNode *root2 = tree2->nodes[tree2->totleaf];

  start_axis = min_axis(tree1->start_axis, tree2->start_axis);
  stop_axis = min_axis(tree1->stop_axis, tree2->stop_axis);

  /* fast check root nodes for collision before doing big splitting + traversal */
  if (!tree_overlap_test(root1, root2, start_axis, stop_axis)) {
    return nullptr;
  }

  data_shared.tree1 = tree1;
  data_shared.tree2 = tree2;
  data_shared.start_axis = start_axis;
  data_shared.stop_axis = stop_axis;

  /* can be NULL */
  data_shared.callback = callback;
  data_shared.userdata = userdata;

  for (j = 0; j < thread_num; j++) {
    /* init BVHOverlapData_Thread */
    data[j].shared = &data_shared;
    data[j].overlap = overlap_pairs ? BLI_stack_new(sizeof(BVHTreeOverlap), __func__) : nullptr;
    data[j].max_interactions = max_interactions;

    /* for callback */
    data[j].thread = j;
  }

  if (use_threading) {
    TaskParallelSettings settings;
    BLI_parallel_range_settings_defaults(&settings);
    settings.min_iter_per_thread = 1;
    BLI_task_parallel_range(0, root_node_len, data, reinterpret_cast<TaskParallelRangeFunc>(bvhtree_overlap_task_cb), &settings);
  }
  else {
    if (max_interactions) {
      tree_overlap_traverse_num(data, root1, root2);
    }
    else if (callback) {
      tree_overlap_traverse_cb(data, root1, root2);
    }
    else {
      tree_overlap_traverse(data, root1, root2);
    }
  }

  if (overlap_pairs) {
    for (j = 0; j < thread_num; j++) {
      total += BLI_stack_count(data[j].overlap);
    }

    to = overlap = static_cast<BVHTreeOverlap*>(MEM_lockfree_mallocN(sizeof(BVHTreeOverlap) * total, "BVHTreeOverlap"));

    for (j = 0; j < thread_num; j++) {
      uint count = (uint)BLI_stack_count(data[j].overlap);
      BLI_stack_pop_n(data[j].overlap, to, count);
      BLI_stack_free(data[j].overlap);
      to += count;
    }
    *r_overlap_tot = (uint)total;
  }

  return overlap;
}

__host__ __device__ BVHTreeOverlap *BLI_bvhtree_overlap(
    const BVHTree *tree1,
    const BVHTree *tree2,
    uint *r_overlap_tot,
    /* optional callback to test the overlap before adding (must be thread-safe!) */
    const BVHTree_OverlapCallback callback,
    void *userdata)
{
  return BLI_bvhtree_overlap_ex(tree1,
                                tree2,
                                r_overlap_tot,
                                callback,
                                userdata,
                                0,
                                BVH_OVERLAP_USE_THREADING | BVH_OVERLAP_RETURN_PAIRS);
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_intersect_plane
 * \{ */

static bool tree_intersect_plane_test(const float *bv, const float plane[4])
{
  /* TODO(germano): Support other kdop geometries. */
  const float bb_min[3] = {bv[0], bv[2], bv[4]};
  const float bb_max[3] = {bv[1], bv[3], bv[5]};
  float bb_near[3], bb_far[3];
  aabb_get_near_far_from_plane(plane, bb_min, bb_max, bb_near, bb_far);
  if ((plane_point_side_v3(plane, bb_near) > 0.0f) !=
      (plane_point_side_v3(plane, bb_far) > 0.0f)) {
    return true;
  }

  return false;
}
//
//static void bvhtree_intersect_plane_dfs_recursive(BVHIntersectPlaneData *__restrict data,
//                                                  const BVHNode *node)
//{
//  if (tree_intersect_plane_test(node->bv, data->plane)) {
//    /* check if node is a leaf */
//    if (!node->totnode) {
//      int *intersect = (int*)BLI_stack_push_r(data->intersect);
//      *intersect = node->index;
//    }
//    else {
//      for (int j = 0; j < data->tree->tree_type; j++) {
//        if (node->children[j]) {
//          bvhtree_intersect_plane_dfs_recursive(data, node->children[j]);
//        }
//      }
//    }
//  }
//}
//
//int *BLI_bvhtree_intersect_plane(BVHTree *tree, float plane[4], uint *r_intersect_tot)
//{
//  int *intersect = nullptr;
//  size_t total = 0;
//
//  if (tree->totleaf) {
//    BVHIntersectPlaneData data;
//    data.tree = tree;
//    copy_v4_v4(data.plane, plane);
//    data.intersect = (BLI_Stack*)BLI_stack_new(sizeof(int), __func__);
//
//    BVHNode *root = tree->nodes[tree->totleaf];
//    bvhtree_intersect_plane_dfs_recursive(&data, root);
//
//    total = BLI_stack_count(data.intersect);
//    if (total) {
//      intersect =(int*)MEM_lockfree_mallocN(sizeof(int) * total, __func__);
//      BLI_stack_pop_n(data.intersect, intersect, (uint)total);
//    }
//    BLI_stack_free(data.intersect);
//  }
//  *r_intersect_tot = (uint)total;
//  return intersect;
//}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_find_nearest
 * \{ */

/* Determines the nearest point of the given node BV.
 * Returns the squared distance to that point. */
static float calc_nearest_point_squared(const float proj[3], const BVHNode *node, float nearest[3])
{
  int i;
  const float *bv = node->bv;

  /* nearest on AABB hull */
  for (i = 0; i != 3; i++, bv += 2) {
    float val = proj[i];
    if (bv[0] > val) {
      val = bv[0];
    }
    if (bv[1] < val) {
      val = bv[1];
    }
    nearest[i] = val;
  }

  return len_squared_v3v3(proj, nearest);
}

/* Depth first search method */
static void dfs_find_nearest_dfs(BVHNearestData *data, BVHNode *node)
{
  if (node->totnode == 0) {
    if (data->callback) {
      data->callback(data->userdata, node->index, data->co, &data->nearest);
    }
    else {
      data->nearest.index = node->index;
      data->nearest.dist_sq = calc_nearest_point_squared(data->proj, node, data->nearest.co);
    }
  }
  else {
    /* Better heuristic to pick the closest node to dive on */
    int i;
    float nearest[3];

    if (data->proj[node->main_axis] <= node->children[0]->bv[node->main_axis * 2 + 1]) {

      for (i = 0; i != node->totnode; i++) {
        if (calc_nearest_point_squared(data->proj, node->children[i], nearest) >=
            data->nearest.dist_sq) {
          continue;
        }
        dfs_find_nearest_dfs(data, node->children[i]);
      }
    }
    else {
      for (i = node->totnode - 1; i >= 0; i--) {
        if (calc_nearest_point_squared(data->proj, node->children[i], nearest) >=
            data->nearest.dist_sq) {
          continue;
        }
        dfs_find_nearest_dfs(data, node->children[i]);
      }
    }
  }
}

static void dfs_find_nearest_begin(BVHNearestData *data, BVHNode *node)
{
  float nearest[3], dist_sq;
  dist_sq = calc_nearest_point_squared(data->proj, node, nearest);
  if (dist_sq >= data->nearest.dist_sq) {
    return;
  }
  dfs_find_nearest_dfs(data, node);
}

/* Priority queue method */
//static void heap_find_nearest_inner(BVHNearestData *data, HeapSimple *heap, BVHNode *node)
//{
//  if (node->totnode == 0) {
//    if (data->callback) {
//      data->callback(data->userdata, node->index, data->co, &data->nearest);
//    }
//    else {
//      data->nearest.index = node->index;
//      data->nearest.dist_sq = calc_nearest_point_squared(data->proj, node, data->nearest.co);
//    }
//  }
//  else {
//    float nearest[3];
//
//    for (int i = 0; i != node->totnode; i++) {
//      float dist_sq = calc_nearest_point_squared(data->proj, node->children[i], nearest);
//
//      //if (dist_sq < data->nearest.dist_sq) {
//      //  BLI_heapsimple_insert(heap, dist_sq, node->children[i]);
//      //}
//    }
//  }
//}

static void heap_find_nearest_begin(const BVHNearestData *data, BVHNode *root)
{
  float nearest[3];
  float dist_sq = calc_nearest_point_squared(data->proj, root, nearest);

  if (dist_sq < data->nearest.dist_sq) {
    //HeapSimple *heap = BLI_heapsimple_new_ex(32);

    //heap_find_nearest_inner(data, heap, root);

    //while (!BLI_heapsimple_is_empty(heap) && BLI_heapsimple_top_value(heap) < data->nearest.dist_sq) 
    //{
    //    BVHNode* node = (BVHNode*)BLI_heapsimple_pop_min(heap);
    //    heap_find_nearest_inner(data, heap, node);
    //}

    //BLI_heapsimple_free(heap, NULL);
  }
}

int BLI_bvhtree_find_nearest_ex(const BVHTree *tree,
                                const float co[3],
                                BVHTreeNearest *nearest,
                                const BVHTree_NearestPointCallback callback,
                                void *userdata,
                                const int flag)
{
  axis_t axis_iter;

  BVHNearestData data;
  BVHNode *root = tree->nodes[tree->totleaf];

  /* init data to search */
  data.tree = tree;
  data.co = co;

  data.callback = callback;
  data.userdata = userdata;

  for (axis_iter = data.tree->start_axis; axis_iter != data.tree->stop_axis; axis_iter++) 
  {
#ifdef __CUDA_ARCH__
      data.proj[axis_iter] = dot_v3v3(data.co, d_bvhtree_kdop_axes[axis_iter]);
#else
      data.proj[axis_iter] = dot_v3v3(data.co, bvhtree_kdop_axes[axis_iter]);
#endif
    
  }

  if (nearest) {
    memcpy(&data.nearest, nearest, sizeof(*nearest));
  }
  else {
    data.nearest.index = -1;
    data.nearest.dist_sq = FLT_MAX;
  }

  /* dfs search */
  if (root) {
    if (flag & BVH_NEAREST_OPTIMAL_ORDER) {
      heap_find_nearest_begin(&data, root);
    }
    else {
      dfs_find_nearest_begin(&data, root);
    }
  }

  /* copy back results */
  if (nearest) {
    memcpy(nearest, &data.nearest, sizeof(*nearest));
  }

  return data.nearest.index;
}

int BLI_bvhtree_find_nearest(BVHTree *tree,
                             const float co[3],
                             BVHTreeNearest *nearest,
                             const BVHTree_NearestPointCallback callback,
                             void *userdata)
{
  return BLI_bvhtree_find_nearest_ex(tree, co, nearest, callback, userdata, 0);
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_find_nearest_first
 * \{ */

static bool isect_aabb_v3(const BVHNode *node, const float co[3])
{
  const BVHTreeAxisRange *bv = (const BVHTreeAxisRange *)node->bv;

  if (co[0] > bv[0].min && co[0] < bv[0].max && co[1] > bv[1].min && co[1] < bv[1].max &&
      co[2] > bv[2].min && co[2] < bv[2].max) {
    return true;
  }

  return false;
}

static bool dfs_find_duplicate_fast_dfs(BVHNearestData *data, BVHNode *node)
{
  if (node->totnode == 0) {
    if (isect_aabb_v3(node, data->co)) {
      if (data->callback) {
        const float dist_sq = data->nearest.dist_sq;
        data->callback(data->userdata, node->index, data->co, &data->nearest);
        return (data->nearest.dist_sq < dist_sq);
      }
      data->nearest.index = node->index;
      return true;
    }
  }
  else {
    /* Better heuristic to pick the closest node to dive on */
    int i;

    if (data->proj[node->main_axis] <= node->children[0]->bv[node->main_axis * 2 + 1]) {
      for (i = 0; i != node->totnode; i++) {
        if (isect_aabb_v3(node->children[i], data->co)) {
          if (dfs_find_duplicate_fast_dfs(data, node->children[i])) {
            return true;
          }
        }
      }
    }
    else {
      for (i = node->totnode; i--;) {
        if (isect_aabb_v3(node->children[i], data->co)) {
          if (dfs_find_duplicate_fast_dfs(data, node->children[i])) {
            return true;
          }
        }
      }
    }
  }
  return false;
}

/**
 * Find the first node nearby.
 * Favors speed over quality since it doesn't find the best target node.
 */
int BLI_bvhtree_find_nearest_first(const BVHTree *tree,
                                   const float co[3],
                                   const float dist_sq,
                                   const BVHTree_NearestPointCallback callback,
                                   void *userdata)
{
  BVHNearestData data;
  BVHNode *root = tree->nodes[tree->totleaf];

  /* init data to search */
  data.tree = tree;
  data.co = co;

  data.callback = callback;
  data.userdata = userdata;
  data.nearest.index = -1;
  data.nearest.dist_sq = dist_sq;

  /* dfs search */
  if (root) {
    dfs_find_duplicate_fast_dfs(&data, root);
  }

  return data.nearest.index;
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_ray_cast
 *
 * raycast is done by performing a DFS on the BVHTree and saving the closest hit.
 *
 * \{ */

/* Determines the distance that the ray must travel to hit the bounding volume of the given node */
static float ray_nearest_hit(const BVHRayCastData *data, const float bv[6])
{
  int i;

  float low = 0, upper = data->hit.dist;

  for (i = 0; i != 3; i++, bv += 2) {
    if (data->ray_dot_axis[i] == 0.0f) {
      /* axis aligned ray */
      if (data->ray.origin[i] < bv[0] - data->ray.radius ||
          data->ray.origin[i] > bv[1] + data->ray.radius) {
        return FLT_MAX;
      }
    }
    else {
      float ll = (bv[0] - data->ray.radius - data->ray.origin[i]) / data->ray_dot_axis[i];
      float lu = (bv[1] + data->ray.radius - data->ray.origin[i]) / data->ray_dot_axis[i];

      if (data->ray_dot_axis[i] > 0.0f) {
        if (ll > low) {
          low = ll;
        }
        if (lu < upper) {
          upper = lu;
        }
      }
      else {
        if (lu > low) {
          low = lu;
        }
        if (ll < upper) {
          upper = ll;
        }
      }

      if (low > upper) {
        return FLT_MAX;
      }
    }
  }
  return low;
}

/**
 * Determines the distance that the ray must travel to hit the bounding volume of the given node
 * Based on Tactical Optimization of Ray/Box Intersection, by Graham Fyffe
 * [http://tog.acm.org/resources/RTNews/html/rtnv21n1.html#art9]
 *
 * TODO this doesn't take data->ray.radius into consideration */
static float fast_ray_nearest_hit(const BVHRayCastData *data, const BVHNode *node)
{
  const float *bv = node->bv;

  float t1x = (bv[data->index[0]] - data->ray.origin[0]) * data->idot_axis[0];
  float t2x = (bv[data->index[1]] - data->ray.origin[0]) * data->idot_axis[0];
  float t1y = (bv[data->index[2]] - data->ray.origin[1]) * data->idot_axis[1];
  float t2y = (bv[data->index[3]] - data->ray.origin[1]) * data->idot_axis[1];
  float t1z = (bv[data->index[4]] - data->ray.origin[2]) * data->idot_axis[2];
  float t2z = (bv[data->index[5]] - data->ray.origin[2]) * data->idot_axis[2];

  if ((t1x > t2y || t2x < t1y || t1x > t2z || t2x < t1z || t1y > t2z || t2y < t1z) ||
      (t2x < 0.0f || t2y < 0.0f || t2z < 0.0f) ||
      (t1x > data->hit.dist || t1y > data->hit.dist || t1z > data->hit.dist)) {
    return FLT_MAX;
  }
  return max_fff(t1x, t1y, t1z);
}

static void dfs_raycast(BVHRayCastData *data, const BVHNode *node)
{
  int i;

  /* ray-bv is really fast.. and simple tests revealed its worth to test it
   * before calling the ray-primitive functions */
  /* XXX: temporary solution for particles until fast_ray_nearest_hit supports ray.radius */
  float dist = (data->ray.radius == 0.0f) ? fast_ray_nearest_hit(data, node) :
                                            ray_nearest_hit(data, node->bv);
  if (dist >= data->hit.dist) {
    return;
  }

  if (node->totnode == 0) {
    if (data->callback) {
      data->callback(data->userdata, node->index, &data->ray, &data->hit);
    }
    else {
      data->hit.index = node->index;
      data->hit.dist = dist;
      madd_v3_v3v3fl(data->hit.co, data->ray.origin, data->ray.direction, dist);
    }
  }
  else {
    /* pick loop direction to dive into the tree (based on ray direction and split axis) */
    if (data->ray_dot_axis[node->main_axis] > 0.0f) {
      for (i = 0; i != node->totnode; i++) {
        dfs_raycast(data, node->children[i]);
      }
    }
    else {
      for (i = node->totnode - 1; i >= 0; i--) {
        dfs_raycast(data, node->children[i]);
      }
    }
  }
}

/**
 * A version of #dfs_raycast with minor changes to reset the index & dist each ray cast.
 */
static void dfs_raycast_all(BVHRayCastData *data, const BVHNode *node)
{
  int i;

  /* ray-bv is really fast.. and simple tests revealed its worth to test it
   * before calling the ray-primitive functions */
  /* XXX: temporary solution for particles until fast_ray_nearest_hit supports ray.radius */
  float dist = (data->ray.radius == 0.0f) ? fast_ray_nearest_hit(data, node) :
                                            ray_nearest_hit(data, node->bv);
  if (dist >= data->hit.dist) {
    return;
  }

  if (node->totnode == 0) {
    /* no need to check for 'data->callback' (using 'all' only makes sense with a callback). */
    dist = data->hit.dist;
    data->callback(data->userdata, node->index, &data->ray, &data->hit);
    data->hit.index = -1;
    data->hit.dist = dist;
  }
  else {
    /* pick loop direction to dive into the tree (based on ray direction and split axis) */
    if (data->ray_dot_axis[node->main_axis] > 0.0f) {
      for (i = 0; i != node->totnode; i++) {
        dfs_raycast_all(data, node->children[i]);
      }
    }
    else {
      for (i = node->totnode - 1; i >= 0; i--) {
        dfs_raycast_all(data, node->children[i]);
      }
    }
  }
}

static void bvhtree_ray_cast_data_precalc(BVHRayCastData *data, int flag)
{
  int i;

  for (i = 0; i < 3; i++) {
#ifdef __CUDA_ARCH__
      data->ray_dot_axis[i] = dot_v3v3(data->ray.direction, d_bvhtree_kdop_axes[i]);
#else
      data->ray_dot_axis[i] = dot_v3v3(data->ray.direction, bvhtree_kdop_axes[i]);
#endif

    if (fabsf(data->ray_dot_axis[i]) < FLT_EPSILON) {
      data->ray_dot_axis[i] = 0.0f;
      /* Sign is not important in this case, `data->index` is adjusted anyway. */
      data->idot_axis[i] = FLT_MAX;
    }
    else {
      data->idot_axis[i] = 1.0f / data->ray_dot_axis[i];
    }

    data->index[2 * i] = data->idot_axis[i] < 0.0f ? 1 : 0;
    data->index[2 * i + 1] = 1 - data->index[2 * i];
    data->index[2 * i] += 2 * i;
    data->index[2 * i + 1] += 2 * i;
  }

#ifdef USE_KDOPBVH_WATERTIGHT
  if (flag & BVH_RAYCAST_WATERTIGHT) {
    isect_ray_tri_watertight_v3_precalc(&data->isect_precalc, data->ray.direction);
    data->ray.isect_precalc = &data->isect_precalc;
  }
  else {
    data->ray.isect_precalc = nullptr;
  }
#else
  UNUSED_VARS(flag);
#endif
}

int BLI_bvhtree_ray_cast_ex(const BVHTree *tree,
                            const float co[3],
                            const float dir[3],
                            const float radius,
                            BVHTreeRayHit *hit,
                            const BVHTree_RayCastCallback callback,
                            void *userdata,
                            const int flag)
{
  BVHRayCastData data;
  BVHNode *root = tree->nodes[tree->totleaf];

  BLI_ASSERT_UNIT_V3(dir);

  data.tree = tree;

  data.callback = callback;
  data.userdata = userdata;

  copy_v3_v3(data.ray.origin, co);
  copy_v3_v3(data.ray.direction, dir);
  data.ray.radius = radius;

  bvhtree_ray_cast_data_precalc(&data, flag);

  if (hit) {
    memcpy(&data.hit, hit, sizeof(*hit));
  }
  else {
    data.hit.index = -1;
    data.hit.dist = BVH_RAYCAST_DIST_MAX;
  }

  if (root) {
    dfs_raycast(&data, root);
    //      iterative_raycast(&data, root);
  }

  if (hit) {
    memcpy(hit, &data.hit, sizeof(*hit));
  }

  return data.hit.index;
}

int BLI_bvhtree_ray_cast(BVHTree *tree,
                         const float co[3],
                         const float dir[3],
                         const float radius,
                         BVHTreeRayHit *hit,
                         const BVHTree_RayCastCallback callback,
                         void *userdata)
{
  return BLI_bvhtree_ray_cast_ex(
      tree, co, dir, radius, hit, callback, userdata, BVH_RAYCAST_DEFAULT);
}

float BLI_bvhtree_bb_raycast(const float bv[6],
                             const float light_start[3],
                             const float light_end[3],
                             float pos[3])
{
  BVHRayCastData data;
  float dist;

  data.hit.dist = BVH_RAYCAST_DIST_MAX;

  /* get light direction */
  sub_v3_v3v3(data.ray.direction, light_end, light_start);

  data.ray.radius = 0.0;

  copy_v3_v3(data.ray.origin, light_start);

  normalize_v3(data.ray.direction);
  copy_v3_v3(data.ray_dot_axis, data.ray.direction);

  dist = ray_nearest_hit(&data, bv);

  madd_v3_v3v3fl(pos, light_start, data.ray.direction, dist);

  return dist;
}

/**
 * Calls the callback for every ray intersection
 *
 * \note Using a \a callback which resets or never sets the #BVHTreeRayHit index & dist works too,
 * however using this function means existing generic callbacks can be used from custom callbacks
 * without having to handle resetting the hit beforehand.
 * It also avoid redundant argument and return value which aren't meaningful
 * when collecting multiple hits.
 */
void BLI_bvhtree_ray_cast_all_ex(const BVHTree *tree,
                                 const float co[3],
                                 const float dir[3],
                                 const float radius,
                                 const float hit_dist,
                                 const BVHTree_RayCastCallback callback,
                                 void *userdata,
                                 const int flag)
{
  BVHRayCastData data;
  BVHNode *root = tree->nodes[tree->totleaf];

  BLI_ASSERT_UNIT_V3(dir);
  BLI_assert(callback != NULL);

  data.tree = tree;

  data.callback = callback;
  data.userdata = userdata;

  copy_v3_v3(data.ray.origin, co);
  copy_v3_v3(data.ray.direction, dir);
  data.ray.radius = radius;

  bvhtree_ray_cast_data_precalc(&data, flag);

  data.hit.index = -1;
  data.hit.dist = hit_dist;

  if (root) {
    dfs_raycast_all(&data, root);
  }
}

void BLI_bvhtree_ray_cast_all(BVHTree *tree,
                              const float co[3],
                              const float dir[3],
                              const float radius,
                              const float hit_dist,
                              const BVHTree_RayCastCallback callback,
                              void *userdata)
{
  BLI_bvhtree_ray_cast_all_ex(
      tree, co, dir, radius, hit_dist, callback, userdata, BVH_RAYCAST_DEFAULT);
}

/** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_range_query
 *
 * Allocates and fills an array with the indices of node that are on the given spherical range
 * (center, radius).
 * Returns the size of the array.
 *
 * \{ */

typedef struct RangeQueryData {
  BVHTree *tree;
  const float *center;
  float radius_sq; /* squared radius */

  int hits;

  BVHTree_RangeQuery callback;
  void *userdata;
} RangeQueryData;

static void dfs_range_query(RangeQueryData *data, BVHNode *node)
{
  if (node->totnode == 0) {
#if 0 /*UNUSED*/
    /* Calculate the node min-coords
     * (if the node was a point then this is the point coordinates) */
    float co[3];
    co[0] = node->bv[0];
    co[1] = node->bv[2];
    co[2] = node->bv[4];
#endif
  }
  else {
    int i;
    for (i = 0; i != node->totnode; i++) {
      float nearest[3];
      float dist_sq = calc_nearest_point_squared(data->center, node->children[i], nearest);
      if (dist_sq < data->radius_sq) {
        /* Its a leaf.. call the callback */
        if (node->children[i]->totnode == 0) {
          data->hits++;
          data->callback(data->userdata, node->children[i]->index, data->center, dist_sq);
        }
        else {
          dfs_range_query(data, node->children[i]);
        }
      }
    }
  }
}

int BLI_bvhtree_range_query(
    BVHTree *tree, const float co[3], const float radius, const BVHTree_RangeQuery callback, void *userdata)
{
  BVHNode *root = tree->nodes[tree->totleaf];

  RangeQueryData data;
  data.tree = tree;
  data.center = co;
  data.radius_sq = radius * radius;
  data.hits = 0;

  data.callback = callback;
  data.userdata = userdata;

  if (root != nullptr) {
    float nearest[3];
    float dist_sq = calc_nearest_point_squared(data.center, root, nearest);
    if (dist_sq < data.radius_sq) {
      /* Its a leaf.. call the callback */
      if (root->totnode == 0) {
        data.hits++;
        data.callback(data.userdata, root->index, co, dist_sq);
      }
      else {
        dfs_range_query(&data, root);
      }
    }
  }

  return data.hits;
}

/** \} */
//
///* -------------------------------------------------------------------- */
///** \name BLI_bvhtree_nearest_projected
// * \{ */
//
//static void bvhtree_nearest_projected_dfs_recursive(BVHNearestProjectedData *__restrict data,
//                                                    const BVHNode *node)
//{
//  if (node->totnode == 0) {
//    if (data->callback) {
//      data->callback(data->userdata, node->index, &data->precalc, nullptr, 0, &data->nearest);
//    }
//    else {
//      data->nearest.index = node->index;
//      data->nearest.dist_sq = dist_squared_to_projected_aabb(
//          &data->precalc,
//          (float[3])(node->bv[0], node->bv[2], node->bv[4]),
//          (float[3])(node->bv[1], node->bv[3], node->bv[5]),
//          data->closest_axis);
//    }
//  }
//  else {
//    /* First pick the closest node to recurse into */
//    if (data->closest_axis[node->main_axis]) {
//      for (int i = 0; i != node->totnode; i++) {
//        const float *bv = node->children[i]->bv;
//
//        if (dist_squared_to_projected_aabb(&data->precalc,
//                                           (float[3])(bv[0], bv[2], bv[4]),
//                                           (float[3])(bv[1], bv[3], bv[5]),
//                                           data->closest_axis) <= data->nearest.dist_sq) {
//          bvhtree_nearest_projected_dfs_recursive(data, node->children[i]);
//        }
//      }
//    }
//    else {
//      for (int i = node->totnode; i--;) {
//        const float *bv = node->children[i]->bv;
//
//        if (dist_squared_to_projected_aabb(&data->precalc,
//                                           (float[3])(bv[0], bv[2], bv[4]),
//                                           (float[3])(bv[1], bv[3], bv[5]),
//                                           data->closest_axis) <= data->nearest.dist_sq) {
//          bvhtree_nearest_projected_dfs_recursive(data, node->children[i]);
//        }
//      }
//    }
//  }
//}
//
//static void bvhtree_nearest_projected_with_clipplane_test_dfs_recursive(
//    BVHNearestProjectedData *__restrict data, const BVHNode *node)
//{
//  if (node->totnode == 0) {
//    if (data->callback) {
//      data->callback(data->userdata,
//                     node->index,
//                     &data->precalc,
//                     data->clip_plane,
//                     data->clip_plane_len,
//                     &data->nearest);
//    }
//    else {
//      data->nearest.index = node->index;
//      data->nearest.dist_sq = dist_squared_to_projected_aabb(
//          &data->precalc,
//          (float[3])(node->bv[0], node->bv[2], node->bv[4]),
//          (float[3])(node->bv[1], node->bv[3], node->bv[5]),
//          data->closest_axis);
//    }
//  }
//  else {
//    /* First pick the closest node to recurse into */
//    if (data->closest_axis[node->main_axis]) {
//      for (int i = 0; i != node->totnode; i++) {
//        const float *bv = node->children[i]->bv;
//        const float bb_min[3] = {bv[0], bv[2], bv[4]};
//        const float bb_max[3] = {bv[1], bv[3], bv[5]};
//
//        int isect_type = isect_aabb_planes_v3(
//            data->clip_plane, data->clip_plane_len, bb_min, bb_max);
//
//        if ((isect_type != ISECT_AABB_PLANE_BEHIND_ANY) &&
//            dist_squared_to_projected_aabb(&data->precalc, bb_min, bb_max, data->closest_axis) <=
//                data->nearest.dist_sq) {
//          if (isect_type == ISECT_AABB_PLANE_CROSS_ANY) {
//            bvhtree_nearest_projected_with_clipplane_test_dfs_recursive(data, node->children[i]);
//          }
//          else {
//            /* ISECT_AABB_PLANE_IN_FRONT_ALL */
//            bvhtree_nearest_projected_dfs_recursive(data, node->children[i]);
//          }
//        }
//      }
//    }
//    else {
//      for (int i = node->totnode; i--;) {
//        const float *bv = node->children[i]->bv;
//        const float bb_min[3] = {bv[0], bv[2], bv[4]};
//        const float bb_max[3] = {bv[1], bv[3], bv[5]};
//
//        int isect_type = isect_aabb_planes_v3(
//            data->clip_plane, data->clip_plane_len, bb_min, bb_max);
//
//        if (isect_type != ISECT_AABB_PLANE_BEHIND_ANY &&
//            dist_squared_to_projected_aabb(&data->precalc, bb_min, bb_max, data->closest_axis) <=
//                data->nearest.dist_sq) {
//          if (isect_type == ISECT_AABB_PLANE_CROSS_ANY) {
//            bvhtree_nearest_projected_with_clipplane_test_dfs_recursive(data, node->children[i]);
//          }
//          else {
//            /* ISECT_AABB_PLANE_IN_FRONT_ALL */
//            bvhtree_nearest_projected_dfs_recursive(data, node->children[i]);
//          }
//        }
//      }
//    }
//  }
//}
//
//int BLI_bvhtree_find_nearest_projected(BVHTree *tree,
//                                       float projmat[4][4],
//                                       float winsize[2],
//                                       float mval[2],
//                                       float clip_plane[6][4],
//                                       int clip_plane_len,
//                                       BVHTreeNearest *nearest,
//                                       BVHTree_NearestProjectedCallback callback,
//                                       void *userdata)
//{
//  BVHNode *root = tree->nodes[tree->totleaf];
//  if (root != nullptr) {
//    BVHNearestProjectedData data;
//    dist_squared_to_projected_aabb_precalc(&data.precalc, projmat, winsize, mval);
//
//    data.callback = callback;
//    data.userdata = userdata;
//
//    if (clip_plane) {
//      data.clip_plane_len = clip_plane_len;
//      for (int i = 0; i < data.clip_plane_len; i++) {
//        copy_v4_v4(data.clip_plane[i], clip_plane[i]);
//      }
//    }
//    else {
//      data.clip_plane_len = 1;
//      planes_from_projmat(projmat, nullptr, nullptr, nullptr, nullptr, data.clip_plane[0], nullptr);
//    }
//
//    if (nearest) {
//      memcpy(&data.nearest, nearest, sizeof(*nearest));
//    }
//    else {
//      data.nearest.index = -1;
//      data.nearest.dist_sq = FLT_MAX;
//    }
//    {
//      const float bb_min[3] = {root->bv[0], root->bv[2], root->bv[4]};
//      const float bb_max[3] = {root->bv[1], root->bv[3], root->bv[5]};
//
//      int isect_type = isect_aabb_planes_v3(data.clip_plane, data.clip_plane_len, bb_min, bb_max);
//
//      if (isect_type != 0 &&
//          dist_squared_to_projected_aabb(&data.precalc, bb_min, bb_max, data.closest_axis) <=
//              data.nearest.dist_sq) {
//        if (isect_type == 1) {
//          bvhtree_nearest_projected_with_clipplane_test_dfs_recursive(&data, root);
//        }
//        else {
//          bvhtree_nearest_projected_dfs_recursive(&data, root);
//        }
//      }
//    }
//
//    if (nearest) {
//      memcpy(nearest, &data.nearest, sizeof(*nearest));
//    }
//
//    return data.nearest.index;
//  }
//  return -1;
//}
//
///** \} */

/* -------------------------------------------------------------------- */
/** \name BLI_bvhtree_walk_dfs
 * \{ */

typedef struct BVHTree_WalkData {
  BVHTree_WalkParentCallback walk_parent_cb;
  BVHTree_WalkLeafCallback walk_leaf_cb;
  BVHTree_WalkOrderCallback walk_order_cb;
  void *userdata;
} BVHTree_WalkData;

/**
 * Runs first among nodes children of the first node before going
 * to the next node in the same layer.
 *
 * \return false to break out of the search early.
 */
static bool bvhtree_walk_dfs_recursive(BVHTree_WalkData *walk_data, const BVHNode *node)
{
  if (node->totnode == 0) {
    return walk_data->walk_leaf_cb(
        (const BVHTreeAxisRange *)node->bv, node->index, walk_data->userdata);
  }

  /* First pick the closest node to recurse into */
  if (walk_data->walk_order_cb(
          (const BVHTreeAxisRange *)node->bv, node->main_axis, walk_data->userdata)) {
    for (int i = 0; i != node->totnode; i++) {
      if (walk_data->walk_parent_cb((const BVHTreeAxisRange *)node->children[i]->bv,
                                    walk_data->userdata)) {
        if (!bvhtree_walk_dfs_recursive(walk_data, node->children[i])) {
          return false;
        }
      }
    }
  }
  else {
    for (int i = node->totnode - 1; i >= 0; i--) {
      if (walk_data->walk_parent_cb((const BVHTreeAxisRange *)node->children[i]->bv,
                                    walk_data->userdata)) {
        if (!bvhtree_walk_dfs_recursive(walk_data, node->children[i])) {
          return false;
        }
      }
    }
  }
  return true;
}

/**
 * This is a generic function to perform a depth first search on the #BVHTree
 * where the search order and nodes traversed depend on callbacks passed in.
 *
 * \param tree: Tree to walk.
 * \param walk_parent_cb: Callback on a parents bound-box to test if it should be traversed.
 * \param walk_leaf_cb: Callback to test leaf nodes, callback must store its own result,
 * returning false exits early.
 * \param walk_order_cb: Callback that indicates which direction to search,
 * either from the node with the lower or higher K-DOP axis value.
 * \param userdata: Argument passed to all callbacks.
 */
void BLI_bvhtree_walk_dfs(const BVHTree *tree,
                          const BVHTree_WalkParentCallback walk_parent_cb,
                          const BVHTree_WalkLeafCallback walk_leaf_cb,
                          const BVHTree_WalkOrderCallback walk_order_cb,
                          void *userdata)
{
  const BVHNode *root = tree->nodes[tree->totleaf];
  if (root != nullptr) {
    BVHTree_WalkData walk_data = {walk_parent_cb, walk_leaf_cb, walk_order_cb, userdata};
    /* first make sure the bv of root passes in the test too */
    if (walk_parent_cb((const BVHTreeAxisRange *)root->bv, userdata)) {
      bvhtree_walk_dfs_recursive(&walk_data, root);
    }
  }
}

/** \} */
